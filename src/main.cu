#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include "common.h"
#include "cusp/csr_matrix.h"


__global__ void generate_groups(unsigned long long int *MatB_bit,
                                unsigned long long int *d_group_mask,
                                int *d_group_ele_row_ind,
                                float *d_group_ele_row_val,
                                float *d_dense,
                                int *group_id,
                                int *spilled_row_cnt,
                                float **tile_spilled_csrVal,
                                int **tile_spilled_csrColInd,
                                int **tile_spilled_csrRowPtr,
                                int *spilled_row_hash_table_reverse_gmem
                                )
{
    int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    int tid = bid * blockDim.x + threadIdx.x;
    int row_ind = blockDim.x * blockIdx.y + threadIdx.x;
    int entry_ind = row_ind * gridDim.x * TILE_WIDTH + blockIdx.x * TILE_WIDTH;
    int entry_ind_bit = row_ind * gridDim.x + blockIdx.x;

    __shared__ unsigned long long int row_group[MAX_GROUP_NUM];
    __shared__ int group_ele_row_idx[MAX_GROUP_NUM][TILE_WIDTH];
    __shared__ float d_dense_smem[SPLIT_K][TILE_WIDTH];
    __shared__ int spilled_row_hash_table_smem[SPLIT_K];
    __shared__ int spilled_row_hash_table_reverse_smem[SPLIT_K];
    // __shared__ int spilled_row_cnt[row_cnt/tile_height*col_cnt/tile_width];

    for (int i = 0; i < SPLIT_K; i++)
    {
        d_dense_smem[threadIdx.x][i] = d_dense[entry_ind + i];
    }

    // Initialize
    if (tid == 0)
    {
        for (int i = 0; i < MAX_GROUP_NUM; i++)
        {
            row_group[i] = 0;
            for (int j = 0; j < TILE_WIDTH; j++)
            {
                group_ele_row_idx[i][j] = -1;
            }
        }
    }

    int group_idx = 0;
    int nnz = 0;
    unsigned long long int and_result; //and_result is used to check if there exists overlap

    unsigned long long int expected = row_group[group_idx];
    // or_result is the group mask after adding to the row_group. In this step, the first group is settled.
    unsigned long long int or_result = row_group[group_idx] | MatB_bit[entry_ind_bit];
    // Only one row is added to the row_group
    unsigned long long int old_value = atomicCAS(&row_group[group_idx], expected, or_result);

    // For rows that haven't been added onto the row_group
    while (expected != old_value) {
        // calculate and_result again to see if there exists overlap
        and_result = row_group[group_idx] & MatB_bit[entry_ind_bit];
        // If there exists overlap, change to next row_group until no overlap exists
        while (and_result != 0) {
            group_idx++;
            if (group_idx >= MAX_GROUP_NUM)
            {
                group_id[entry_ind_bit] = -1;
                int spilled_row_hash_key = atomicAdd(&spilled_row_cnt[bid], 1);
                spilled_row_hash_table_smem[spilled_row_hash_key] = threadIdx.x;
                for (int j = 0; j < TILE_WIDTH; j++)
                {
                    if (d_dense_smem[threadIdx.x][j] != 0.0f)
                    {
                        atomicAdd(&nnz, 1);
                    }
                }
                return;
            }
            and_result = row_group[group_idx] & MatB_bit[entry_ind_bit];
        }
        expected = row_group[group_idx];
        // Now there is no overlap, try to add onto the new row_group.
        or_result = row_group[group_idx] | MatB_bit[entry_ind_bit];
        old_value = atomicCAS(&row_group[group_idx], expected, or_result);
    }

    for (int i = 0; i < TILE_WIDTH; i++) {
        if ((MatB_bit[entry_ind_bit] >> i & 1) == 0x01) {
            group_ele_row_idx[group_idx][i] = threadIdx.x;
        }
    }

    __syncthreads();
    if (threadIdx.x == 0)
    {
        int nz_ind = 0;
        int spilled_row;
        hipMalloc((void**) &tile_spilled_csrColInd[bid], nnz * sizeof(int));
        hipMalloc((void**) &tile_spilled_csrVal[bid], nnz * sizeof(float));
        hipMalloc((void**) &tile_spilled_csrRowPtr[bid], (spilled_row_cnt[bid]+1) * sizeof(int));
        tile_spilled_csrRowPtr[bid][0] = 0;
        for (int i = 0; i < spilled_row_cnt[bid]; i++)
        {
            spilled_row = spilled_row_hash_table_smem[i];
            spilled_row_hash_table_reverse_smem[spilled_row] = i;
            for (int j = 0; j < TILE_WIDTH; j++)
            {
                if (d_dense_smem[spilled_row][j] != 0.0f)
                {
                    tile_spilled_csrColInd[bid][nz_ind] = j;
                    tile_spilled_csrVal[bid][nz_ind] = d_dense_smem[spilled_row][j];
                    nz_ind++;
                }
            }
            tile_spilled_csrRowPtr[bid][i+1] = nz_ind;
        }
    }

    // Load the csr information back to global memory
    spilled_row_hash_table_reverse_gmem[bid * SPLIT_K + threadIdx.x] 
                = spilled_row_hash_table_reverse_smem[threadIdx.x];

    // load the group information into global memory
    for (int i = 0; i < MAX_GROUP_NUM; i++)
    {
        d_group_mask[MAX_GROUP_NUM * bid + i] = row_group[i];
    }
    for (int i = 0; i < TILE_WIDTH; i++) {
        d_group_ele_row_ind[(MAX_GROUP_NUM * bid + group_idx) * TILE_WIDTH + i] 
                = group_ele_row_idx[group_idx][i];
        d_group_ele_row_val[(MAX_GROUP_NUM * bid + group_idx) * TILE_WIDTH + i] 
                = d_dense_smem[group_ele_row_idx[group_idx][i]][i];
    }
    group_id[entry_ind_bit] = group_idx;
}

// __device__ void ld_groups_to_regs(int *d_group_ele_row_idx, 
//                                   unsigned long long int *d_group_mask,
//                                   float *dB_dense
//                                     )
// {
//     // int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     // column per thread
//     // Load rows to groups and // add up groups into registers
//     for (int i = 0; i < MAX_GROUP_NUM; i++) {
//         // Need modification, should read from the CSR or other condensed formats.
//         group[i][tid] = dB_dense[d_group_ele_row_idx[i][tid]][tid];
//     }
    
//     // MAX_GROUP_NUM = 4
//     group[4][tid] = group[0][tid] + group[1][tid];
//     group[5][tid] = group[0][tid] + group[2][tid];
//     group[6][tid] = group[0][tid] + group[3][tid];
//     group[7][tid] = group[1][tid] + group[2][tid];
//     group[8][tid] = group[1][tid] + group[3][tid];
//     group[9][tid] = group[2][tid] + group[3][tid];
//     group[10][tid] = group[4][tid] + group[2][tid];
//     group[11][tid] = group[4][tid] + group[3][tid];
//     group[12][tid] = group[5][tid] + group[3][tid];
//     group[13][tid] = group[7][tid] + group[3][tid];
//     group[14][tid] = group[10][tid] + group[3][tid];
//     group[15][tid] = group[4][tid] + group[9][tid];

// }

__global__ void bit_wise_spgemm(int split_k,
                                float *d_csr_values, 
                                int *d_csr_offsets, 
                                int *d_csr_columns,
                                float *d_group_ele_row_val,
                                unsigned long long int *MatB_bit,           // MatrixB's bit mask
                                int *group_id_gmem,                         // MatrixB's group ID
                                int *spilled_row_cnt,
                                float **tile_spilled_csrVal,
                                int **tile_spilled_csrColInd,
                                int **tile_spilled_csrRowPtr,
                                int *spilled_row_hash_table_reverse_gmem
                                )
{
    int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    // int tid = bid * blockDim.x + threadIdx.x;

    int assigned_row_ind = blockIdx.y * blockDim.x + threadIdx.x;
    // int assigned_col_ind = blockIdx.x * split_k;
    // int assigned_bit_pos = threadIdx.x % BIT_WIDTH;
    int entry_ind_bit = assigned_row_ind * gridDim.x + blockIdx.x;

    int row_ind_in_tile, row_group_id, register_idx, col_ind;
    __shared__ unsigned long long int group_indicator[TILE_HEIGHT][BIT_WIDTH][MAX_GROUP_NUM];
    __shared__ float result[TILE_HEIGHT][BIT_WIDTH][TILE_WIDTH];
    __shared__ int group_id_smem[SPLIT_K];
    __shared__ int spilled_row_hash_table_reverse_smem[SPLIT_K];

    for (int k = 0; k < SIZE_K/SPLIT_K; k++)
    {
        // Load group_id to shared memory
        // Load spilled_row_hash_table_reverse to shared memory
        for (int i = 0; i < SPLIT_K/blockDim.x; i++)
        {
            int row_ind = k * SPLIT_K + i * blockDim.x + threadIdx.x;
            int entry = row_ind * gridDim.x + blockIdx.x;
            group_id_smem[i * blockDim.x + threadIdx.x] = group_id_gmem[entry];
            spilled_row_hash_table_reverse_smem[i * blockDim.x + threadIdx.x] 
                = spilled_row_hash_table_reverse_gmem[entry];
        }

        // Transform an specific area of the CSR of MatrixA to a tiled form.
        // The transformation process is inherited in this kernel
        for (int i = d_csr_offsets[assigned_row_ind]; i < d_csr_offsets[assigned_row_ind+1]; i++)
        {
            if (d_csr_columns[i] > SPLIT_K * k && d_csr_columns[i] < SPLIT_K * (k+1))
            {
                row_ind_in_tile = d_csr_columns[i] - SPLIT_K * k;
                for (int b = 0; b < BIT_WIDTH; b++)
                {
                    if((__float_as_int(d_csr_values[i]) >> b) & 1 == 0x01)
                    {
                        row_group_id = group_id_smem[row_ind_in_tile];
                        if (row_group_id != -1)
                        {
                            int entry = ((TILE_HEIGHT * k) + row_ind_in_tile) * gridDim.x + blockIdx.x;
                            group_indicator[threadIdx.x][b][row_group_id] 
                                = atomicOr(&group_indicator[threadIdx.x][b][row_group_id], MatB_bit[entry]);
                        }
                        else 
                        {
                            // Current row_ind_in_tile in MatrixB is the spilled row
                            // Perform the extra computation
                            int row_in_csr = spilled_row_hash_table_reverse_smem[row_ind_in_tile];
                            int tileB_id = k * gridDim.x + blockIdx.x;
                            for (int j = tile_spilled_csrRowPtr[tileB_id][row_in_csr]; 
                                    j < tile_spilled_csrRowPtr[tileB_id][row_in_csr+1]; j++)
                            {
                                col_ind = tile_spilled_csrColInd[tileB_id][j];
                                result[threadIdx.x][b][col_ind] += tile_spilled_csrVal[tileB_id][col_ind];
                            }
                        }
                    }
                }
            }
        }
    }


    float group[MAX_GROUP_NUM];
    // Load groups to registers, one column per thread
#pragma unroll
    for (int i = 0; i < MAX_GROUP_NUM; i++)
    {
        group[1 << i] = d_group_ele_row_val[(MAX_GROUP_NUM * bid + i) * 64 + threadIdx.x];
    }
    // Calculate the combinations of groups and store the results in registers if possible
    // ...


    // One column per thread to read values from registers
    // To achieve so, the tile_width and the tile_height should be the same
    for (int i = 0; i < blockDim.x; i++) 
    {
        for (int b = 0; b < BIT_WIDTH; b++)
        {
            for (int j = 0; j < MAX_GROUP_NUM; j++) 
            {
                // register_idx = atomicAnd(&register_idx, (group_indicator[i][b][j] << threadIdx.x)); // this is one bit
                if (group_indicator[i][b][j] == 1)
                {
                    register_idx = group_indicator[i][b][j] << threadIdx.x;
                    result[i][b][threadIdx.x] += group[register_idx];
                }
            }
            // result[i][b][tid] = group[register_idx];
        }
    }

}

__global__ void dense2bitmask(float *MatB_dense, unsigned long long int *MatB_bit)
{
    // int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    // int tid = bid * blockDim.x + threadIdx.x;

    int row_ind = blockDim.x * blockIdx.y + threadIdx.x;
    int col_ind = blockIdx.x * 64;
    int entry_ind = row_ind * gridDim.x * 64 + col_ind;
    int entry_ind_bit = row_ind * gridDim.x + blockIdx.x;
    for (int i = 0; i < 64; i++)
    {
        if (MatB_dense[entry_ind + i] != 0.0f)
        {
            atomicOr(&MatB_bit[entry_ind_bit], ((unsigned long long int)1 << i));
        }
    }
}

int dense2CSR(int num_rows, 
                int num_cols, 
                float *d_dense, 
                float *d_csr_values, 
                int *d_csr_offsets, 
                int *d_csr_columns)
{
    int ld = num_cols;
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix B in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, num_rows, num_cols, 0,
                                      d_csr_offsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &d_csr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_csr_values,  nnz * sizeof(float)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns,
                                           d_csr_values) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    return 0;
}



int main() 
{
    // using IndexType = int;
    // using ValueType = float;
    // using CSRHost = cusp::csr_matrix<IndexType,ValueType,cusp::host_memory>;
    // using CSRDev = cusp::csr_matrix<IndexType,ValueType,cusp::device_memory>;

    const int m = SIZE_M;
    const int k = SIZE_K;
	const int n = SIZE_N;

    int sparsity = 90;
	float* hA_dense = (float*)malloc(sizeof(float)*m*k);
    float* hB_dense = (float*)malloc(sizeof(float)*k*n);
    float* hC_dense = (float*)malloc(sizeof(float)*m*n);
    fill_random(hA_dense, m, k, sparsity);
    fill_random(hB_dense, k, n, sparsity);
    fill_random(hC_dense, m, n, sparsity);

    float *dA_dense, *dA_csr_values, *dB_dense, *dB_group_ele_val;
    int   *dA_csr_offsets, *dA_csr_columns, *dB_group_id, *dB_spilled_row_cnt, *dB_spilled_row_hash_table_reverse_gmem;
    unsigned long long int *dB_bitmask, *dB_groupmask;
    int *dB_group_ele_ind;
    float **dB_tile_spilled_csrVal;
    int **dB_tile_spilled_csrColInd, **dB_tile_spilled_csrRowPtr;

    CHECK_CUDA( hipMalloc((void**) &dA_dense,          m * k * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dA_csr_offsets,   (m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_dense,          k * n * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB_bitmask,        k * n / TILE_WIDTH * sizeof(unsigned long long int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_groupmask,      k * n / TILE_HEIGHT / TILE_WIDTH * MAX_GROUP_NUM * sizeof(unsigned long long int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_ele_ind,  k * n / TILE_HEIGHT * MAX_GROUP_NUM * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_ele_val,  k * n / TILE_HEIGHT * MAX_GROUP_NUM * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_id,       k * n / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_cnt,k * n / TILE_HEIGHT / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_hash_table_reverse_gmem,
                                    k * n / TILE_HEIGHT / TILE_WIDTH * TILE_HEIGHT * sizeof(int)) )

    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrVal,     k * n / TILE_HEIGHT / TILE_WIDTH * sizeof(float*)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrColInd,  k * n / TILE_HEIGHT / TILE_WIDTH * sizeof(int*)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrRowPtr,  k * n / TILE_HEIGHT / TILE_WIDTH * sizeof(int*)) )
    
    CHECK_CUDA( hipMemcpy(dA_dense, hA_dense, m * k * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_dense, hB_dense, k * n * sizeof(float),
                           hipMemcpyHostToDevice) )

    dim3 grid(SIZE_K/TILE_HEIGHT, SIZE_N/TILE_WIDTH, 1), block(TILE_HEIGHT, 1, 1);

    printf("Matrix B dense2bitmask...\n");
    dense2bitmask<<<grid, block>>>(dB_dense, dB_bitmask);

    printf("Matrix A dense2CSR...\n");
    dense2CSR(m, k, dA_dense, dA_csr_values, dA_csr_offsets, dA_csr_columns);

    unsigned long long int *hB_bitmask = (unsigned long long int*)malloc(sizeof(unsigned long long int)*k*n/64);
    hipMemcpy(hB_bitmask, dB_bitmask, k * n / 64 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    printlongintMatrix(k, n/64, hB_bitmask, "B_bitmask");
    // printfloatMatrix(k, n, hB_dense, "MatB");

    printf("\nMatrix B generate groups...\n");
    generate_groups<<<grid, block>>>(dB_bitmask,                            // input
                                     dB_groupmask,                          // output, for visualization
                                     dB_group_ele_ind,                      // output, not necessary
                                     dB_group_ele_val,                      // output
                                     dB_dense,                              // input
                                     dB_group_id,                           // output
                                     dB_spilled_row_cnt,                    // output
                                     dB_tile_spilled_csrVal,                // output
                                     dB_tile_spilled_csrColInd,             // output
                                     dB_tile_spilled_csrRowPtr,             // output
                                     dB_spilled_row_hash_table_reverse_gmem // output
                                     );

    // spgemm
    bit_wise_spgemm<<<grid, block>>>(SPLIT_K, 
                                    dA_csr_values, 
                                    dA_csr_offsets, 
                                    dA_csr_columns, 
                                    dB_group_ele_val, 
                                    dB_bitmask, 
                                    dB_group_id, 
                                    dB_spilled_row_cnt, 
                                    dB_tile_spilled_csrVal, 
                                    dB_tile_spilled_csrColInd, 
                                    dB_tile_spilled_csrRowPtr, 
                                    dB_spilled_row_hash_table_reverse_gmem
                                    );


    unsigned long long int *hB_groupmask = (unsigned long long int*)malloc(k * n / TILE_HEIGHT / TILE_WIDTH * MAX_GROUP_NUM * sizeof(unsigned long long int));
    int *hB_group_ele_ind = (int*)malloc(k * n / TILE_HEIGHT * MAX_GROUP_NUM * sizeof(int));
    hipMemcpy(hB_groupmask, dB_groupmask, k * n / TILE_HEIGHT / TILE_WIDTH * MAX_GROUP_NUM * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(hB_group_ele_ind, dB_group_ele_ind, k * n / TILE_HEIGHT * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);

    printlongintMatrix(k, n/64, hB_groupmask, "B_groupmask");
    std::cout << "A random number: " << rand() % 100 << std::endl;
    for (int i = 0; i < 64; i++)
    {
        std::cout << std::left << std::setw(4) << hB_group_ele_ind[i];
    }

    // free(dB)

    // std::cout << "Input matrix A has shape (" << A.num_rows << "," << A.num_cols << ") and " << A.num_entries << " entries" << "\n";
    // std::cout << "             B has shape (" << B.num_rows << "," << B.num_cols << ") and " << B.num_entries << " entries" << "\n\n";

}