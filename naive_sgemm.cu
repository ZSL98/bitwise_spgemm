#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>

void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}

bool check(const float *A,
           const float *B,
           const float *C,
           int m, int n, int k) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.f;
            for (int p = 0; p < k; ++p) {
                sum += A[i * k + p] * B[j + p * n];
            }

            if (std::fabs(sum - C[i * n + j]) / std::fabs(sum) > 1e-5f) {
                printf("C[%d][%d] not match, %f vs %f\n", i, j, sum, C[i * n + j]);
                return false;
            }
        }
    }

    return true;
}





int main() {
    int m = 5120;
    int n = 4096;
    int k = 4096;
    int n_iter = 10;

    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, m * k * sizeof(float));
    hipHostMalloc(&h_B, k * n * sizeof(float));
    hipHostMalloc(&h_C, m * n * sizeof(float));
    random_init(h_A, m * k);
    random_init(h_B, k * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyDefault);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 grid((n + 63) / 64, (m + 63) / 64, (k + 3) / 4);
    dim3 block(64);

    int *h_A_ctl, *d_A_ctl;
    hipHostMalloc(&h_A_ctl, 64 * 32 * 3 * sizeof(int));
    hipMemset(&h_A_ctl, 0, 64 * 32 * 3);
    hipMalloc(&d_A_ctl, 64 * 32 * 3 * sizeof(int));
    hipMemcpy(d_A_ctl, h_A_ctl, 64 * 32 * 3 * sizeof(int), hipMemcpyDefault);

    // warmup
    bit_wise_gemm_s<<<grid, block>>>(
        m, n, k, d_A_ctl, d_B, d_C);

    hipEventRecord(start);
    for (int i = 0; i < n_iter; ++i) {
        bit_wise_gemm_s<<<grid, block>>>(
            m, n, k, d_A_ctl, d_B, d_C);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    long workload = n_iter * long(m) * n * k * 2;
    double gflops = (double(workload) / 1e9) / (double(ms) / 1e3);
    printf("Performance: %fGFLOPS\n", gflops);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDefault);

    bool chk = check(h_A, h_B, h_C, m, n, k);
    printf("Matrix_C check: %s\n", chk ? "OK" : "Failed");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
}
