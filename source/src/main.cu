#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <cusparseLt.h>

#include "common.h"
#include "transform.cuh"
#include "cuda_utils.cuh"
#include "utils.h"
#include "cusp/csr_matrix.h"
#include "cusp/timer.h"

// tsparse include
#include <thrust/host_vector.h>
#include <thrust/find.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <thrust/functional.h> //bit_or
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/count.h>
#include <thrust/extrema.h>
#include <thrust/transform_reduce.h>
#include <thrust/system/omp/execution_policy.h>

#include <algorithm> //find
#include <vector>

#include "mm.h"

// include TileSpGEMM
// #include "TileSpGEMM/common.h"
// #include "TileSpGEMM/mmio_highlevel.h"
// #include "TileSpGEMM/utils.h"
// #include "TileSpGEMM/utils_cuda_scan.h"
// #include "TileSpGEMM/spgemm_nsparse_kernel.h"
#include "TileSpGEMM/csr2tile.h"
#include "TileSpGEMM/tilespgemm-cuda.h"
// #include "TileSpGEMM/spgemm-cpu.h"
// #include "TileSpGEMM/tile2csr.h"
// #include "TileSpGEMM/spgemm_serialref_spa_new.h"
// #include "TileSpGEMM/spgemm_cu.h"



template <typename BitMaskType,
          typename InitValueType,
          typename ValueType>
__global__ void generate_groups(BitMaskType *MatB_bit,
                                BitMaskType *d_group_mask,
                                // int *d_group_ele_row_ind,
                                ValueType *d_group_ele_row_val,
                                InitValueType *d_dense,
                                int *group_id,
                                int *spilled_row_cnt,
                                int *spilled_nnz,
                                // float **tile_spilled_csrVal,
                                // int **tile_spilled_csrColInd,
                                // int **tile_spilled_csrRowPtr,
                                int *spilled_row_hash_table_gmem,
                                int *spilled_row_hash_table_reverse_gmem,
                                int *nnz
                                )
{
    int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    int row_ind = blockDim.x * blockIdx.y + threadIdx.x;
    int entry_ind = row_ind * gridDim.x * TILE_WIDTH + blockIdx.x * TILE_WIDTH;
    int entry_ind_bit = row_ind * gridDim.x + blockIdx.x;

    __shared__ int row_group[MAX_GROUP_NUM];
    __shared__ int group_ele_row_idx[MAX_GROUP_NUM][TILE_WIDTH];
    __shared__ InitValueType d_dense_smem[SPLIT_K][TILE_WIDTH];
    __shared__ int spilled_row_hash_table_smem[SPLIT_K];
    __shared__ int spilled_row_hash_table_reverse_smem[SPLIT_K];
    // __shared__ int spilled_row_cnt[row_cnt/tile_height*col_cnt/tile_width];

    spilled_row_hash_table_smem[threadIdx.x] = 0;
    spilled_row_hash_table_reverse_smem[threadIdx.x] = -1;
    for (int i = 0; i < TILE_WIDTH; i++)
    {
        d_dense_smem[threadIdx.x][i] = d_dense[entry_ind + i];
    }

    // Initialize
    if (threadIdx.x == 0)
    {
        for (int i = 0; i < MAX_GROUP_NUM; i++)
        {
            row_group[i] = 0;
            for (int j = 0; j < TILE_WIDTH; j++)
            {
                group_ele_row_idx[i][j] = -1;
            }
        }
    }

    int mask = MatB_bit[entry_ind_bit];
    __syncthreads();

    int group_idx = 0;
    if (mask == 0)
    {
        group_idx = 0;
    }
    else
    {
        BitMaskType and_result; //and_result is used to check if there exists overlap
        BitMaskType expected = row_group[group_idx];
        and_result = expected & mask;
        while (and_result != 0)
        {
            // if (bid == 0)
            // {
            //     printf("Collision. Move to next.\n");
            // }
            group_idx++;
            expected = row_group[group_idx];
            and_result = expected & mask;
        }

        // BitMaskType expected = row_group[group_idx];
        // or_result is the group mask after adding to the row_group. In this step, the first group is settled.
        BitMaskType or_result = expected | mask;
        // Only one row is added to the row_group
        BitMaskType old_value = atomicCAS(&row_group[group_idx], expected, or_result);

        // For rows that haven't been added onto the row_group
        while (expected != old_value) {
            // if (bid == 0)
            // {
            //     printf("Not stored: %d, group_idx: %d, thread: %d\n", mask, group_idx, threadIdx.x);
            // }
            // calculate and_result again to see if there exists overlap
            expected = row_group[group_idx];
            and_result = expected & mask;
            // If there exists overlap, change to next row_group until no overlap exists
            while (and_result != 0) {
                // if (bid == 0)
                // {
                //     printf("Collision. Move to next again.\n");
                // }
                group_idx++;
                if (group_idx >= MAX_GROUP_NUM)
                {
                    group_id[entry_ind_bit] = -1;
                    int spilled_row_hash_key = atomicAdd(&spilled_row_cnt[bid], 1);
                    spilled_row_hash_table_smem[spilled_row_hash_key] = threadIdx.x;
                    for (int j = 0; j < TILE_WIDTH; j++)
                    {
                        if (d_dense_smem[threadIdx.x][j] != 0)
                        {
                            atomicAdd(&spilled_nnz[bid], 1);
                        }
                    }
                    break;
                }
                expected = row_group[group_idx];
                and_result = expected & mask;
            }
            if (group_idx >= MAX_GROUP_NUM)
            {
                break;
            }
            // expected = row_group[group_idx];
            // Now there is no overlap, try to add onto the new row_group.
            or_result = expected | mask;
            old_value = atomicCAS(&row_group[group_idx], expected, or_result);
            // printf("Bid: %d, thread: %d, group_idx: %d\n", bid, threadIdx.x, group_idx);
        }
    }
    // row_group[group_idx] |= MatB_bit[entry_ind_bit];

    group_id[entry_ind_bit] = group_idx;

    // if (bid == 0)
    // {
    //     printf("thread: %d, group_idx: %d, bitmask: %d\n", threadIdx.x, group_idx, MatB_bit[entry_ind_bit]);
    // }

    for (int i = 0; i < TILE_WIDTH; i++) {
        if (mask >> (31-i) & 1) {
            group_ele_row_idx[group_idx][i] = threadIdx.x;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < MAX_GROUP_NUM; i++)
        {
            atomicAdd(nnz, __popc(row_group[i]));
        }
        int spilled_row;

        for (int i = 0; i < spilled_row_cnt[bid]; i++)
        {
            spilled_row = spilled_row_hash_table_smem[i];
            spilled_row_hash_table_reverse_smem[spilled_row] = i;
        }

        // dB_tile_spilled_csrVal[bid] = (float*)malloc(spilled_nnz[bid]);
        // dB_tile_spilled_csrColInd[bid] = (int*)malloc(spilled_nnz[bid]);
        // dB_tile_spilled_csrRowPtr[bid] = (int*)malloc(spilled_row_cnt[bid]+1);

        // printf("bid: %d, nnz: %d\n", bid, spilled_nnz[bid]);

        // load the group information into global memory
        for (int i = 0; i < MAX_GROUP_NUM; i++)
        {
            d_group_mask[MAX_GROUP_NUM * bid + i] = row_group[i];
        }
        for (int g = 0; g < MAX_GROUP_NUM; g++)
        {
            for (int i = 0; i < TILE_WIDTH; i++) {
                // d_group_ele_row_ind[(MAX_GROUP_NUM * bid + group_idx) * TILE_WIDTH + i] 
                //         = group_ele_row_idx[group_idx][i];
                if(group_ele_row_idx[g][i] >=0)
                {
                    d_group_ele_row_val[(MAX_GROUP_NUM * bid + g) * TILE_WIDTH + i] 
                            = (ValueType)d_dense_smem[group_ele_row_idx[g][i]][i];
                }
            }
        }
        // group_id[entry_ind_bit] = group_idx;
    }
    __syncthreads();
    // Load the csr information back to global memory
    spilled_row_hash_table_reverse_gmem[bid * SPLIT_K + threadIdx.x] 
                = spilled_row_hash_table_reverse_smem[threadIdx.x];
    spilled_row_hash_table_gmem[bid * SPLIT_K + threadIdx.x] 
                = spilled_row_hash_table_smem[threadIdx.x];
    // __syncthreads();

}


template <typename BitMaskType,
          typename InitValueType,
          typename ValueType>
__global__ void generate_spilled_csr(BitMaskType *MatB_bit,
                                InitValueType *d_dense,
                                int *group_id,
                                int *spilled_row_cnt,
                                int *spilled_nnz,
                                int *spilled_row_cnt_offset,
                                int *spilled_nnz_offset,
                                ValueType *tile_spilled_csrVal,
                                int *tile_spilled_csrColInd,
                                int *tile_spilled_csrRowPtr,
                                int *spilled_row_hash_table_gmem,
                                int *spilled_row_hash_table_reverse_gmem
                                )
{

    int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    int row_ind = blockDim.x * blockIdx.y + threadIdx.x;
    int entry_ind = row_ind * gridDim.x * TILE_WIDTH + blockIdx.x * TILE_WIDTH;
    // int entry_ind_bit = row_ind * gridDim.x + blockIdx.x;

    __shared__ InitValueType d_dense_smem[SPLIT_K][TILE_WIDTH];
    __shared__ int spilled_row_hash_table_smem[SPLIT_K];
    for (int i = 0; i < TILE_WIDTH; i++)
    {
        d_dense_smem[threadIdx.x][i] = d_dense[entry_ind + i];
    }
    spilled_row_hash_table_smem[threadIdx.x] = spilled_row_hash_table_gmem[bid * SPLIT_K + threadIdx.x];
    
    if (threadIdx.x == 0)
    {
        int nz_ind_total = 0;
        int row_ind_total = 0;
        int spilled_row;

        // tile_spilled_csrRowPtr[0] = 0;
        for (int i = 0; i < spilled_row_cnt[bid]; i++)
        {
            spilled_row = spilled_row_hash_table_smem[i];
            for (int j = 0; j < TILE_WIDTH; j++)
            {
                if (d_dense_smem[spilled_row][j] != 0)
                {
                    tile_spilled_csrColInd[spilled_nnz_offset[bid] + nz_ind_total] = j;
                    tile_spilled_csrVal[spilled_nnz_offset[bid] + nz_ind_total] = (ValueType)d_dense_smem[spilled_row][j];
                    nz_ind_total++;
                }
            }
            tile_spilled_csrRowPtr[spilled_row_cnt_offset[bid] + row_ind_total] = nz_ind_total;
            row_ind_total++;
        }
    }
}

template <typename InitValueType,
          typename ValueType>
__global__ void csr2tiledcsr(
                int tileA_cnt,
                int64_t dA_nnz,
                int *dA_csr_offset,
                int *dA_csr_column,
                InitValueType *dA_csr_value,
                int *tiled_csr_offset,
                int *tiled_csr_column,
                ValueType *tiled_csr_value,
                int *tile_nnz_acc,
                int *tile_nnz,
                int *tile_row_nnz
                )
{
    // __shared__ int tile_row_nnz[SIZE_M][SIZE_K/SPLIT_K];
    for (int i = 0; i < SIZE_M+1; i++)
    {
        int start_offset = dA_csr_offset[i];
        int end_offset = dA_csr_offset[i+1];
        for (int j = start_offset; j < end_offset; j++)
        {
            int tileA_y = i / TILE_HEIGHT;
            int tileA_x = dA_csr_column[j] / SPLIT_K;
            int tileA_id = tileA_y * (SIZE_K / SPLIT_K) + tileA_x;
            tile_nnz[tileA_id]++;
        }
    }

    int tmp_cnt = 0;
    for (int i = 0; i < tileA_cnt; i++)
    {
        tile_nnz_acc[i] = tmp_cnt;
        tmp_cnt += tile_nnz[i];
    }
    tile_nnz_acc[tileA_cnt] = tmp_cnt;

    int tmp_tile_nnz[(SIZE_M/TILE_HEIGHT)*(SIZE_K/SPLIT_K)];

    for (int i = 0; i < (SIZE_M/TILE_HEIGHT)*(SIZE_K/SPLIT_K); i++)
    {
        tmp_tile_nnz[i] = 0;
    }

    for (int i = 0; i < SIZE_M+1; i++)
    {
        int start_offset = dA_csr_offset[i];
        int end_offset = dA_csr_offset[i+1];
        for (int j = start_offset; j < end_offset; j++)
        {
            int tileA_y = i / TILE_HEIGHT;
            int tileA_x = dA_csr_column[j] / SPLIT_K;
            int tileA_id = tileA_y * (SIZE_K / SPLIT_K) + tileA_x;
            tile_row_nnz[i * (SIZE_K / SPLIT_K) + tileA_x]++;

            // int tile_entry_y = i % TILE_HEIGHT;
            int tile_entry_x = dA_csr_column[j] % SPLIT_K;
            // int tile_entry = tile_entry_y * SPLIT_K + tile_entry_x;

            int tile_offset = tile_nnz_acc[tileA_id];
            int entry = tile_offset + tmp_tile_nnz[tileA_id];
            tmp_tile_nnz[tileA_id]++;

            // printf("entry: %d\n", entry);
            tiled_csr_value[entry] = (ValueType)dA_csr_value[j];
            tiled_csr_column[entry] = tile_entry_x;
        }
    }

    for (int i = 0; i < tileA_cnt; i++)
    {
        int tileA_y = i / (SIZE_K/SPLIT_K);
        int tileA_x = i % (SIZE_K/SPLIT_K);
        int tile_nnz_tmp = 0;
        for (int j = 0; j < TILE_HEIGHT+1; j++)
        {
            tiled_csr_offset[(TILE_HEIGHT+1)*i + j] = tile_nnz_tmp; 
            tile_nnz_tmp += tile_row_nnz[(tileA_y*TILE_HEIGHT+j) * (SIZE_K / SPLIT_K) + tileA_x];
        } 
    }
}


template <typename BitMaskType,
          typename ValueType>
__global__ void dense2bitmask(ValueType *MatB_dense, BitMaskType *MatB_bit)
{
    int bid = blockIdx.y * gridDim.x + blockIdx.x;  
    int row_ind = blockDim.x * blockIdx.y + threadIdx.x;
    int entry_ind = row_ind * gridDim.x * TILE_WIDTH + blockIdx.x * TILE_WIDTH;
    int entry_ind_bit = row_ind * gridDim.x + blockIdx.x;
    if (TILE_WIDTH == 64) 
    {
        for (int i = 0; i < 64; i++)
        {
            if (MatB_dense[entry_ind + i] != 0)
            {
                atomicOr(&MatB_bit[entry_ind_bit], ((unsigned long long int)1 << (63-i)));
            }
        }
    }
    else if (TILE_WIDTH == 32)
    {
        for (int i = 0; i < 32; i++)
        {
            if (MatB_dense[entry_ind + i] != 0)
            {
                // if (bid == 0)
                // {
                //     printf("MatB_dense: %f\n", MatB_dense[entry_ind + i]);
                // }
                atomicOr(&MatB_bit[entry_ind_bit], (1 << (31-i)));
                // if (bid == 0)
                // {
                //     printf("MatB_bit: %d, entry_ind_bit: %d, i: %d, MatB_dense: %f\n", MatB_bit[entry_ind_bit], entry_ind_bit, i, MatB_dense[entry_ind + i]);
                // }
            }
        }
    }
    __syncthreads();
    // if (bid == 0)
    // {
    //     printf("thread: %d, entry_ind_bit: %d, bitmask: %d\n", threadIdx.x, entry_ind_bit, MatB_bit[entry_ind_bit]);
    // }
}

int dense2CSR(int num_rows, 
                int num_cols, 
                float *&d_dense, 
                float *&d_csr_values, 
                int *&d_csr_offsets, 
                int *&d_csr_columns,
                int64_t &nnzA
                )
{
    int ld = num_cols;
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix B in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, num_rows, num_cols, 0,
                                      d_csr_offsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )


    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                         &nnzA) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &d_csr_columns, nnzA * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_csr_values,  nnzA * sizeof(float)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns,
                                           d_csr_values) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    return 0;
}

template <typename ValueType>
int coo2bmp_noTuple_d(const cusp::coo_matrix<int, ValueType, cusp::device_memory>& in,
    cusp::coo_matrix<int, uint64_t, cusp::device_memory>& out,
    thrust::device_vector<ValueType>& elems, thrust::device_vector<uint32_t>& idx) {

    using IndexType = int;
    using ElemIndexType = uint32_t;
    using UnsignedIndexType = uint32_t;
    using LongIndexType = uint64_t;
    using BMPType = uint64_t;
    // using ValueType = float;
    using ValueTypeBMP = uint64_t;
    using COOHostBMP = cusp::coo_matrix<IndexType,ValueTypeBMP,cusp::host_memory>;
    using COODevBMP =  cusp::coo_matrix<IndexType,ValueTypeBMP,cusp::device_memory>;
    using COOHost =    cusp::coo_matrix<IndexType,ValueType,cusp::host_memory>;
    using COODev =     cusp::coo_matrix<IndexType,ValueType,cusp::device_memory>;

    auto exec = thrust::cuda::par;

    COODev in_copy(in);

    // sort COO first, it is needed for elem_array. The COO matrix gets sorted by row, and each row by column.
    thrust::sort_by_key(exec, in_copy.column_indices.begin(), in_copy.column_indices.end(), thrust::make_zip_iterator(
            thrust::make_tuple(in_copy.row_indices.begin(), in_copy.values.begin())));
    thrust::stable_sort_by_key(exec, in_copy.row_indices.begin(), in_copy.row_indices.end(), thrust::make_zip_iterator(
            thrust::make_tuple(in_copy.column_indices.begin(), in_copy.values.begin())));

    thrust::device_vector<LongIndexType> tile_indices(in_copy.num_entries); //Absolute index of the tile each element belongs to
    thrust::device_vector<BMPType> position(in_copy.num_entries); //Absolute index of each element inside respective tile (1<<index)

    // Finds 2 things. a) In which tile each element belongs. Tile is returned with absolute indexing. b) What is the
    // position of each element in the respective tile. The position is returned with absolute indexing.
    thrust::for_each(exec,
            thrust::make_zip_iterator(
                    thrust::make_tuple(in_copy.row_indices.begin(), in_copy.column_indices.begin(), tile_indices.begin(),
                            position.begin())),
            thrust::make_zip_iterator(
                    thrust::make_tuple(in_copy.row_indices.end(), in_copy.column_indices.end(), tile_indices.end(), position.end())),
            find_tile_index(in_copy.num_cols));

    // Sort row_indices, col_indices, values and positions in tile by the absolute index of the tile. The sort is stable
    // in order to keep the order of values (elements). The values are expected to come from a COO matrix that has the
    // rows ordered and the columns of each row ordered.
    thrust::stable_sort_by_key(exec, tile_indices.begin(), tile_indices.end(),
            thrust::make_zip_iterator(
                    thrust::make_tuple(in_copy.row_indices.begin(), in_copy.column_indices.begin(), in_copy.values.begin(),
                            position.begin())));

    thrust::device_vector<LongIndexType> tile_indices_unique(in_copy.num_entries); //Unique absolute indices of tiles
    thrust::device_vector<BMPType> bmp(in_copy.num_entries);

    thrust::equal_to<UnsignedIndexType> binary_pred;
    thrust::bit_or<BMPType> binary_op;
    // Elements are reduced based on the index of the tile they belong to. This function returns the unique tile indices and the
    // the result of reduction is the total bmp of all elements that belong to the same tile.
    auto new_end = thrust::reduce_by_key(exec, tile_indices.begin(), tile_indices.end(), position.begin(), tile_indices_unique.begin(),
            bmp.begin(), binary_pred, binary_op);

    UnsignedIndexType num_of_tiles = new_end.first - tile_indices_unique.begin();

    idx.resize(num_of_tiles);

    // transform BMP to population counts
    thrust::transform(exec, bmp.begin(), new_end.second, idx.begin(), bmp_popcount_d());

    // convert population counts to offsets
    thrust::exclusive_scan(exec, idx.begin(), idx.end(), idx.begin(), UnsignedIndexType(0));

    out.num_rows = in_copy.num_rows / BMP_DIM  + ((in_copy.num_rows % BMP_DIM)?1:0) ;
    out.num_cols = in_copy.num_cols / BMP_DIM  + ((in_copy.num_cols % BMP_DIM)?1:0) ;
    out.num_entries = num_of_tiles;
    out.resize(out.num_rows, out.num_cols, out.num_entries);

    // Convert absolute tile indices to relative indexing, to be stored in the COO matrix of the output
    thrust::for_each(exec,
            thrust::make_zip_iterator(
                    thrust::make_tuple(tile_indices_unique.begin(), out.row_indices.begin(), out.column_indices.begin())),
            thrust::make_zip_iterator(
                    thrust::make_tuple(new_end.first, out.row_indices.end(), out.column_indices.end())),
            absolute2relative(out.num_cols));

    thrust::copy(bmp.begin(), new_end.second, out.values.begin());

    elems.resize(in_copy.num_entries);
    thrust::copy(in_copy.values.begin(), in_copy.values.end(), elems.begin());

    return 1;
}


template <typename InputType>
float time_spmmBMP_noTuple(const InputType& A_h, const InputType& B_h)
{
    using IndexType = int;
    using ValueType = signed char;
    using ValueTypeBMP = uint64_t;
    using ElemIndexType = uint32_t;
    using COODevBMP = cusp::coo_matrix<IndexType,ValueTypeBMP,cusp::device_memory>;
    using COOHost   = cusp::coo_matrix<IndexType,ValueType,cusp::host_memory>;
    using COODev    = cusp::coo_matrix<IndexType,ValueType,cusp::device_memory>;

    unsigned int N = REPETITIONS; //repetitions for timing
    const COOHost A_COO_h(A_h), B_COO_h(B_h);

    COODev A_COO_d(A_COO_h);
    COODev B_COO_d(B_COO_h);

    COODevBMP A_BMP_d;
    COODevBMP B_BMP_d;

    thrust::device_vector<ValueType> A_elems_d;
    thrust::device_vector<ValueType> B_elems_d;
    thrust::device_vector<ValueType> C_elems_d; //This is initialized inside the multiply routine

    thrust::device_vector<ElemIndexType> A_idx_d;
    thrust::device_vector<ElemIndexType> B_idx_d;
    thrust::device_vector<ElemIndexType> C_idx_d; //This is initialized inside the multiply routine

    timer t_conv;
    coo2bmp_noTuple_d(A_COO_d, A_BMP_d, A_elems_d, A_idx_d);
    coo2bmp_noTuple_d(B_COO_d, B_BMP_d, B_elems_d, B_idx_d);
    float time_conversion = t_conv.milliseconds_elapsed();
    // printf(" COO to bitmap conversion (for both inputs) time: %lfms\n", time_conversion);

    timer t;

    for(unsigned int i = 0; i < N; i++)
    {
        COODevBMP C_BMP_d;
        thrust::device_vector<ValueType> C_elems_d;
        multiplyBmp_noTuple(A_BMP_d, A_elems_d, A_idx_d, B_BMP_d, B_elems_d, B_idx_d, C_BMP_d, C_elems_d, C_idx_d);
    }

    float time_elapsed = t.milliseconds_elapsed() / N;
    return time_elapsed;
}

template <typename OutputType,
          typename BitMaskType>
__global__ void group2dense(OutputType *d_group_value, OutputType *d_dense, int *d_output_group_idx, BitMaskType *d_bitmask)
{
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    int tid = bid * blockDim.x + threadIdx.x;

    BitMaskType bitmask = d_bitmask[tid];
    int group_id = d_output_group_idx[tid];
    if (group_id >= 0)
    {
        for (int i = 0; i < TILE_WIDTH; i++)
        {
            if (d_bitmask[tid] >> (31 - i) & 0x01)
            {
                int in_entry = bid * OUTPUT_MAX_GROUP_NUM * TILE_WIDTH + group_id * TILE_WIDTH + i;
                int out_entry = (blockIdx.y * blockDim.x + threadIdx.x) * SIZE_N + blockIdx.x * TILE_WIDTH + i;
                d_dense[out_entry] = d_group_value[in_entry];
            }
        }
    }
}


template <typename OutputType,
          typename BitMaskType>
__global__ void dense2group_from_idx(OutputType *d_dense, OutputType *d_group_value, int *d_output_group_idx, BitMaskType *d_bitmask)
{
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    int tid = bid * blockDim.x + threadIdx.x;

    int group_id = d_output_group_idx[tid];

    if (tid == 4)
    {
        printf("\ntid: %d, group_id: %d, bitmask: %d\n", tid, group_id, d_bitmask[tid]);
    }
    if (group_id >= 0)
    {
        for (int i = 0; i < TILE_WIDTH; i++)
        {
            if (tid == 4)
            {
                printf("%d", d_bitmask[tid] >> (31 - i) & 0x01);
            }
            if (d_bitmask[tid] >> (31 - i) & 0x01)
            {
                int in_entry = bid * OUTPUT_MAX_GROUP_NUM * TILE_WIDTH + group_id * TILE_WIDTH + i;
                int out_entry = (blockIdx.y * blockDim.x + threadIdx.x) * SIZE_N + blockIdx.x * TILE_WIDTH + i;
                d_group_value[in_entry] = d_dense[out_entry];
            }
        }
        if (tid == 4)
        {
            printf("\n");
        }
    }
}


void initialize_multiplicand(half *h_multiplicand)
{
    for (int i = 0 ; i < 8*16; i++)
    {
        h_multiplicand[i] = 0;
    }
    h_multiplicand[0] = 1;
    h_multiplicand[1] = 2;
    h_multiplicand[2] = 4;
    h_multiplicand[3] = 8;
    h_multiplicand[4] = 16;
    h_multiplicand[5] = 32;
    h_multiplicand[6] = 64;
    h_multiplicand[7] = -128;
    h_multiplicand[24] = 1;
    h_multiplicand[25] = 2;
    h_multiplicand[26] = 4;
    h_multiplicand[27] = 8;
    h_multiplicand[28] = 16;
    h_multiplicand[29] = 32;
    h_multiplicand[30] = 64;
    h_multiplicand[31] = -128;

}

void initialize_diag_multiplicand(half *h_multiplicand)
{
    for (int i = 0 ; i < 8*16; i++)
    {
        h_multiplicand[i] = 0;
    }
    h_multiplicand[0] = 1;
    h_multiplicand[17] = 1;
    h_multiplicand[17*2] = 1;
    h_multiplicand[17*3] = 1;
    h_multiplicand[17*4] = 1;
    h_multiplicand[17*5] = 1;
    h_multiplicand[17*6] = 1;
    h_multiplicand[17*7] = 1;

}

void initialize_SMatrix(SMatrix *&matrix, int row_size, int col_size, int64_t nnz, 
                  int *&csrRowPtr, int *&csrColIdx, float *&csrVal)
{
    matrix->m = row_size;
    matrix->n = col_size;
    matrix->nnz = nnz;
    matrix->rowpointer = csrRowPtr;
    matrix->columnindex = csrColIdx;
    matrix->value = csrVal;

}

float timing_cusparse_spgemm(int64_t &nnzA, int64_t &nnzB, int64_t &nnzC,
                             int *&dA_csr_offsets, 
                             int *&dA_csr_columns, 
                             float *&dA_csr_values,

                             int *&dB_csr_offsets, 
                             int *&dB_csr_columns, 
                             float *&dB_csr_values,

                             int   *&dC_csrOffsets, 
                             int *&dC_columns,
                             float *&dC_values
                             )
{
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, SIZE_M, SIZE_K, nnzA,
                                      dA_csr_offsets, dA_csr_columns, dA_csr_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, SIZE_K, SIZE_N, nnzB,
                                      dB_csr_offsets, dB_csr_columns, dB_csr_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, SIZE_M, SIZE_N, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    //--------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseHandle_t     handle = NULL;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    hipEventRecord(start);
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )

    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &nnzC) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, nnzC * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  nnzC * sizeof(float)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    hipEventRecord(end);
    hipEventSynchronize(end);

    float cusparse_ms;
    hipEventElapsedTime(&cusparse_ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    return cusparse_ms;
}


float timing_cusparse_spmm_csr(int64_t &nnzA,
                             int *&dA_csr_offsets, 
                             int *&dA_csr_columns, 
                             float *&dA_csr_values,

                             float *&dB_dense
                             )
{
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseHandle_t     handle = NULL;
    void*                dBuffer    = NULL;
    size_t bufferSize = 0;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB_spmm, matC_spmm;
    float                *dC_dense;
    CHECK_CUDA( hipMalloc((void**) &dC_dense, SIZE_M * SIZE_N * sizeof(float)))

    hipDeviceSynchronize();

    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, SIZE_M, SIZE_K, nnzA,
                                      dA_csr_offsets, dA_csr_columns, dA_csr_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB_spmm, SIZE_K, SIZE_N, SIZE_N, dB_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC_spmm, SIZE_M, SIZE_N, SIZE_N, dC_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )

    hipEventRecord(start);
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB_spmm, &beta, matC_spmm, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB_spmm, &beta, matC_spmm, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);


    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB_spmm) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC_spmm) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    hipFree(dBuffer);
    hipFree(dC_dense);

    return time_ms;
}

template <typename InputType>
float timing_cusparseLt(InputType *&dA, 
                        InputType *&dB
                             )
{
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    int major_cc, minor_cc;
    CHECK_CUDA( hipDeviceGetAttribute(&major_cc,
                                       hipDeviceAttributeComputeCapabilityMajor, 0) )
    CHECK_CUDA( hipDeviceGetAttribute(&minor_cc,
                                       hipDeviceAttributeComputeCapabilityMinor, 0) )
    if (!(major_cc == 8 && minor_cc == 0) &&
        !(major_cc == 8 && minor_cc == 6)) {
        std::printf("\ncusparseLt is supported only on GPU devices with"
                    " compute capability == 8.0, 8.6 current: %d.%d\n\n",
                     major_cc, minor_cc);
        return -1;
    }

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    constexpr int m     = SIZE_M; // bigger sizes may require dynamic allocations
    constexpr int n     = SIZE_K; // bigger sizes may require dynamic allocations
    constexpr int k     = SIZE_N; // bigger sizes may require dynamic allocations
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB   = HIPSPARSE_OPERATION_TRANSPOSE;
    auto          type  = HIP_R_8I;
    auto          compute_type = CUSPARSE_COMPUTE_32I;
    if (typeid(InputType) == typeid(__half)) 
    {
        printf("cusparseLt using fp16\n");
        type  = HIP_R_16F;
        compute_type = CUSPARSE_COMPUTE_16F;
    }
    else if (typeid(InputType) == typeid(signed char))
    {
        printf("cusparseLt using int8\n");
        type  = HIP_R_8I;
        compute_type = CUSPARSE_COMPUTE_32I;
    }

    bool     is_rowmajor    = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows     = (isA_transposed) ? k : m;
    auto     num_A_cols     = (isA_transposed) ? m : k;
    auto     num_B_rows     = (isB_transposed) ? n : k;
    auto     num_B_cols     = (isB_transposed) ? k : n;
    auto     num_C_rows     = m;
    auto     num_C_cols     = n;
    unsigned alignment      = 16;
    auto     lda            = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb            = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc            = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height       = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height       = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height       = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size         = A_height * lda * sizeof(InputType);
    auto     B_size         = B_height * ldb * sizeof(InputType);
    auto     C_size         = C_height * ldc * sizeof(int);


    InputType *dA_compressed;
    int    *dC, *dD;
    int    *d_valid;
    CHECK_CUDA( hipMalloc((void**) &dC, C_size) )
    CHECK_CUDA( hipMalloc((void**) &d_valid, sizeof(d_valid)) )
    dD = dC;

    //--------------------------------------------------------------------------
    // cusparseLt data structures and handle initialization
    cusparseLtHandle_t             handle;
    cusparseLtMatDescriptor_t      matA, matB, matC;
    cusparseLtMatmulDescriptor_t   matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t         plan;
    hipStream_t                   stream = nullptr;
    CHECK_CUSPARSE( cusparseLtInit(&handle) )
    // matrix descriptor initialization
    CHECK_CUSPARSE( cusparseLtStructuredDescriptorInit(
                                            &handle, &matA, num_A_rows,
                                            num_A_cols, lda, alignment,
                                            type, order,
                                            CUSPARSELT_SPARSITY_50_PERCENT) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matB, num_B_rows,
                                            num_B_cols, ldb, alignment,
                                            type, order) )
    CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                            &handle, &matC, num_C_rows,
                                            num_C_cols, ldc, alignment,
                                            type, order) )
    // matmul, algorithm selection, and plan initialization
    CHECK_CUSPARSE( cusparseLtMatmulDescriptorInit(
                                            &handle, &matmul, opA, opB,
                                            &matA, &matB, &matC, &matC,
                                            compute_type) )
    CHECK_CUSPARSE( cusparseLtMatmulAlgSelectionInit(
                                            &handle, &alg_sel, &matmul,
                                            CUSPARSELT_MATMUL_ALG_DEFAULT) )
    int alg = 0;
    CHECK_CUSPARSE( cusparseLtMatmulAlgSetAttribute(
                                            &handle, &alg_sel,
                                            CUSPARSELT_MATMUL_ALG_CONFIG_ID,
                                            &alg, sizeof(alg)))
    size_t workspace_size, compressed_size;
    CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel,
                                             workspace_size) )

    CHECK_CUSPARSE( cusparseLtMatmulGetWorkspace(&handle, &plan,
                                                 &workspace_size))

    //--------------------------------------------------------------------------
    // Prune the A matrix (in-place) and check the correcteness
    CHECK_CUSPARSE( cusparseLtSpMMAPrune(&handle, &matmul, dA, dA,
                                         CUSPARSELT_PRUNE_SPMMA_TILE, stream) )
    CHECK_CUSPARSE( cusparseLtSpMMAPruneCheck(&handle, &matmul, dA,
                                              d_valid, stream) )
    int is_valid;
    CHECK_CUDA( hipMemcpyAsync(&is_valid, d_valid, sizeof(d_valid),
                                hipMemcpyDeviceToHost, stream) )
    CHECK_CUDA( hipStreamSynchronize(stream) )
    if (is_valid != 0) {
        std::printf("!!!! The matrix has been pruned in a wrong way. "
                    "cusparseLtMatmul will not provide correct results\n");
        return EXIT_FAILURE;
    }
    //--------------------------------------------------------------------------
    // Compress the A matrix
    CHECK_CUSPARSE( cusparseLtSpMMACompressedSize(&handle, &plan,
                                                  &compressed_size) )
    CHECK_CUDA( hipMalloc((void**) &dA_compressed, compressed_size) )

    CHECK_CUSPARSE( cusparseLtSpMMACompress(&handle, &plan, dA,
                                            dA_compressed, stream) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Search the best kernel
    void*         d_workspace = nullptr;
    int           num_streams = 0;
    hipStream_t* streams     = nullptr;
    // CHECK_CUSPARSE( cusparseLtMatmulSearch(&handle, &plan, &alpha,
    //                                        dA_compressed, dB, &beta,
    //                                        dC, dD, d_workspace,
    //                                        streams, num_streams) )
    int alg_id;
    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(
                                           &handle, &alg_sel,
                                           CUSPARSELT_MATMUL_ALG_CONFIG_ID,
                                           &alg_id, sizeof(alg_id)) )
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Perform the matrix multiplication
    hipEventRecord(start);

    CHECK_CUSPARSE( cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB,
                                     &beta, dC, dD, d_workspace, streams,
                                     num_streams) )


    hipEventRecord(end);
    hipEventSynchronize(end);

    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // destroy plan and handle
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matA) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matB) )
    CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matC) )
    CHECK_CUSPARSE( cusparseLtMatmulPlanDestroy(&plan) )
    CHECK_CUSPARSE( cusparseLtDestroy(&handle) )
    return time_ms;
}


int bitspgemm_prepare(int64_t &nnzA, int64_t &nnzB, int64_t &nnzC,
                        float *&dA_csr_values, 
                        int *&dA_csr_offsets, 
                        int *&dA_csr_columns,

                        int *&dA_tiled_csr_offset,
                        int *&dA_tiled_csr_column,
                        ValueType *&dA_tiled_csr_value,
                        int *&dA_tile_nnz_acc,
                        int *&dA_tile_nnz,
                        int *&dA_tile_row_nnz,

                        float *&dB_dense,

                        BitMaskType *&dB_bitmask,
                        BitMaskType *&dB_groupmask,
                        ValueType *&dB_group_ele_val,
                        int *&dB_group_id,
                        int *&dB_spilled_row_cnt,
                        int *&dB_spilled_nnz,
                        int *&dB_spilled_row_hash_table_gmem,
                        int *&dB_spilled_row_hash_table_reverse_gmem,

                        ValueType *&dB_tile_spilled_csrVal,
                        int *&dB_tile_spilled_csrColInd, 
                        int *&dB_tile_spilled_csrRowPtr,
                        int *&dB_spilled_nnz_offset, 
                        int *&dB_spilled_row_cnt_offset
                      )
{
    int tileA_cnt = (SIZE_M/TILE_HEIGHT)*(SIZE_K/SPLIT_K);
    int tileB_cnt = SIZE_K * SIZE_N / SPLIT_K / TILE_WIDTH;
    int tileB_x_cnt = SIZE_N / TILE_WIDTH;
    int tileB_y_cnt = SIZE_K / SPLIT_K;

    //--------------------------------------------------------------------------
    // Matrix A transformation
    printf("Transform CSR to tiled CSR\n");

    CHECK_CUDA( hipMalloc((void**) &dA_tile_nnz,         sizeof(int) * tileA_cnt) )
    CHECK_CUDA( hipMalloc((void**) &dA_tile_nnz_acc,     sizeof(int) * (tileA_cnt+1)) )
    CHECK_CUDA( hipMalloc((void**) &dA_tile_row_nnz,     sizeof(int) * SIZE_M * SIZE_K / SPLIT_K) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_offset, sizeof(int) * tileA_cnt * (TILE_HEIGHT+1)) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_column, sizeof(int) * nnzA) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_value,  sizeof(ValueType) * nnzA) )

    csr2tiledcsr<<<1, 1>>>(tileA_cnt, 
                            nnzA, 
                            dA_csr_offsets, 
                            dA_csr_columns, 
                            dA_csr_values,
                            dA_tiled_csr_offset,
                            dA_tiled_csr_column,
                            dA_tiled_csr_value,
                            dA_tile_nnz_acc,
                            dA_tile_nnz,
                            dA_tile_row_nnz
                            );

    //--------------------------------------------------------------------------
    // Matrix B transformation

    dim3 grid1(tileB_x_cnt, tileB_y_cnt, 1), block1(SPLIT_K, 1, 1);
    printf("Matrix B dense2bitmask...\n");
    dense2bitmask<<<grid1, block1>>>(dB_dense, dB_bitmask);

    int *dB_nnz;
    CHECK_CUDA( hipMalloc((void**) &dB_nnz, sizeof(int) * 1) )
    printf("Matrix B generate groups...\n");
    generate_groups<<<grid1, block1>>>(dB_bitmask,                            // input
                                     dB_groupmask,                          // output, for visualization
                                    //  dB_group_ele_ind,                      // output, not necessary
                                     dB_group_ele_val,                      // output
                                     dB_dense,                              // input
                                     dB_group_id,                           // output
                                     dB_spilled_row_cnt,                    // output
                                     dB_spilled_nnz,
                                    //  dB_tile_spilled_csrVal,                // output
                                    //  dB_tile_spilled_csrColInd,             // output
                                    //  dB_tile_spilled_csrRowPtr,             // output
                                     dB_spilled_row_hash_table_gmem,
                                     dB_spilled_row_hash_table_reverse_gmem,   // output
                                     dB_nnz
                                     );
                                     
    int *hB_spilled_nnz = (int*)malloc(tileB_cnt * sizeof(int));
    int *hB_spilled_row_cnt = (int*)malloc(tileB_cnt * sizeof(int));
    hipMemcpy(hB_spilled_nnz, dB_spilled_nnz, tileB_cnt * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hB_spilled_row_cnt, dB_spilled_row_cnt, tileB_cnt * sizeof(int), hipMemcpyDeviceToHost);

    int nnz_cnt = 0;
    int row_cnt = 0;
    int *hB_spilled_nnz_offset = (int*)malloc(tileB_cnt * sizeof(int));
    int *hB_spilled_row_cnt_offset = (int*)malloc(tileB_cnt * sizeof(int));
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_nnz_offset,     tileB_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_cnt_offset,  tileB_cnt * sizeof(int)) )
    for (int i = 0; i < tileB_cnt; i++)
    {
        hB_spilled_nnz_offset[i] = nnz_cnt;
        hB_spilled_row_cnt_offset[i] = row_cnt;
        nnz_cnt += hB_spilled_nnz[i];
        row_cnt += hB_spilled_row_cnt[i];
    }
    hipMemcpy(dB_spilled_nnz_offset, hB_spilled_nnz_offset, tileB_cnt * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB_spilled_row_cnt_offset, hB_spilled_row_cnt_offset, tileB_cnt * sizeof(int), hipMemcpyHostToDevice);

    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrVal,     nnz_cnt * sizeof(ValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrColInd,  nnz_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrRowPtr,  row_cnt * sizeof(int)) )

    generate_spilled_csr<<<grid1, block1>>>(dB_bitmask,                            // input
                                     dB_dense,                              // input
                                     dB_group_id,                           // output
                                     dB_spilled_row_cnt,                    // output
                                     dB_spilled_nnz,
                                     dB_spilled_row_cnt_offset,
                                     dB_spilled_nnz_offset,
                                     dB_tile_spilled_csrVal,                // output
                                     dB_tile_spilled_csrColInd,             // output
                                     dB_tile_spilled_csrRowPtr,             // output
                                     dB_spilled_row_hash_table_gmem,
                                     dB_spilled_row_hash_table_reverse_gmem // output
                                     );

    std::cout << "Total nnz: " << nnz_cnt << "  Total row_cnt: " << row_cnt << std::endl;
    CHECK_CUDA(hipDeviceSynchronize())

    return 0;
}

int timing_bitspgemm(int64_t &nnzA, int64_t &nnzB, int64_t &nnzC,

                        int *&dA_tiled_csr_offset,
                        int *&dA_tiled_csr_column,
                        ValueType *&dA_tiled_csr_value,
                        int *&dA_tile_nnz_acc,
                        int *&dA_tile_nnz,
                        int *&dA_tile_row_nnz,

                        float *&dB_dense,

                        BitMaskType *&dB_bitmask,
                        BitMaskType *&dB_groupmask,
                        ValueType *&dB_group_ele_val,
                        int *&dB_group_id,
                        int *&dB_spilled_row_cnt,
                        int *&dB_spilled_nnz,
                        int *&dB_spilled_row_hash_table_gmem,
                        int *&dB_spilled_row_hash_table_reverse_gmem,

                        ValueType *&dB_tile_spilled_csrVal,
                        int *&dB_tile_spilled_csrColInd, 
                        int *&dB_tile_spilled_csrRowPtr,
                        int *&dB_spilled_nnz_offset, 
                        int *&dB_spilled_row_cnt_offset,
                        
                        int *&dC_output_group_idx,
                        BitMaskType *&dC_bitmask,
                        float *&dC_group_value
                        )
{

    int tileB_cnt = SIZE_K * SIZE_N / SPLIT_K / TILE_WIDTH;
    int tileC_cnt = SIZE_M * SIZE_N / TILE_HEIGHT / TILE_WIDTH;

    half *d_multiplicand;
    CHECK_CUDA( hipMalloc((void**) &d_multiplicand,  8 * 16 * sizeof(half)) )
    half *h_multiplicand = (half*)malloc(8 * 16 * sizeof(half));
    initialize_multiplicand(h_multiplicand);
    hipMemcpy(d_multiplicand, h_multiplicand, 8 * 16 * sizeof(half), hipMemcpyHostToDevice);


    BitMaskType *dC_groupmask;
    int *dC_spilled_row_cnt, *dC_spilled_nnz;
    int *dC_spilled_row_row_idx, *dC_spilled_row_tile_idx;
    CHECK_CUDA( hipMalloc((void**) &dC_group_value,  tileC_cnt * (OUTPUT_MAX_GROUP_NUM*4) * TILE_WIDTH * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_bitmask,  SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_groupmask,  tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_cnt,  tileC_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz,  tileC_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_output_group_idx,  SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_row_idx,  MAX_SPILLED_ROW_CNT_C * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_tile_idx,  MAX_SPILLED_ROW_CNT_C * sizeof(int)) )

    int *dC_spilled_row_buffersize, *dC_spilled_nnz_buffersize;
    int *dC_spilled_nnz_offset, *dC_spilled_row_cnt_offset;
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz_offset,     (tileC_cnt + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_cnt_offset,  (tileC_cnt + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_buffersize,  sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz_buffersize,  sizeof(int)) )

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // spgemm
    hipEventRecord(start);
    
    int *dC_nnz;
    CHECK_CUDA( hipMalloc((void**) &dC_nnz, sizeof(int) * 1) )
    int *hC_nnz = (int*)malloc(sizeof(int));

    dim3 grid_2d(SIZE_N/TILE_WIDTH, SIZE_M/TILE_HEIGHT, 1), block_1d(TILE_HEIGHT, 1, 1);
    pre_spgemm<<<grid_2d, block_1d>>>(dB_bitmask, 
                                      dC_spilled_row_cnt, 
                                      dC_spilled_nnz, 
                                      dA_tiled_csr_offset,
                                      dA_tiled_csr_column,  
                                      dA_tile_nnz_acc, 
                                      dC_output_group_idx,
                                      dC_bitmask,
                                      dC_groupmask,
                                      dC_spilled_row_row_idx,
                                      dC_spilled_row_tile_idx,
                                      dC_spilled_row_cnt_offset,
                                      dC_spilled_nnz_offset,
                                      dC_spilled_row_buffersize,
                                      dC_spilled_nnz_buffersize,
                                      dC_nnz
                                      );

    hipEventRecord(end);
    hipEventSynchronize(end);

    // hipMemcpy(hC_nnz, dC_nnz, sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "nnz of C in groups: " << *hC_nnz << std::endl; 

    // int* hC_output_group_idx = (int*) malloc(SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int));
    // hipMemcpy(hC_output_group_idx, dC_output_group_idx, SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int), hipMemcpyDeviceToHost);
    // printf("\n hC_output_group_idx: %d\n", hC_output_group_idx[0]);
    // printMatrix(16, 16, hC_output_group_idx, "hC_output_group_idx");

    BitMaskType* hC_groupmask = (BitMaskType*)malloc(tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType));
    hipMemcpy(hC_groupmask, dC_groupmask, tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType), hipMemcpyDeviceToHost);
    printf("\n hC_groupmask: %d\n", hC_groupmask[0]);
    printintMatrix_32(16, hC_groupmask, "hC_groupmask");

    // int* hC_bitmask = (int*)malloc(SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int));
    // hipMemcpy(hC_bitmask, dC_bitmask, SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int), hipMemcpyDeviceToHost);
    // printf("\n hC_bitmask: %d\n", hC_bitmask[4]);
    // printintMatrix_32(16, hC_bitmask, "hC_bitmask");

    // printf("pre_spgemm success!\n");
    // int *hC_spilled_row_buffersize = (int*)malloc(sizeof(int));
    // int *hC_spilled_nnz_buffersize = (int*)malloc(sizeof(int));
    // hipMemcpy(hC_spilled_row_buffersize, dC_spilled_row_buffersize, sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(hC_spilled_nnz_buffersize, dC_spilled_nnz_buffersize, sizeof(int), hipMemcpyDeviceToHost);


    // int *dC_tile_spilled_csrRowPtr, *dC_tile_spilled_csrColInd;
    // float *dC_tile_spilled_csrVal;
    // CHECK_CUDA( hipMalloc((void**) &dC_tile_spilled_csrRowPtr,  *hC_spilled_row_buffersize * sizeof(int)) )
    // CHECK_CUDA( hipMalloc((void**) &dC_tile_spilled_csrColInd,  *hC_spilled_nnz_buffersize * sizeof(int)) )
    // CHECK_CUDA( hipMalloc((void**) &dC_tile_spilled_csrVal,     *hC_spilled_nnz_buffersize * sizeof(float)) )

    // if (*hC_spilled_row_buffersize != 0)
    // {
    //     spgemm_compute_spilled<<<1, *hC_spilled_row_buffersize>>>(
    //                                     dC_spilled_row_row_idx,
    //                                     dC_spilled_row_tile_idx,
    //                                     dA_tiled_csr_offset,
    //                                     dA_tiled_csr_column,
    //                                     dA_tiled_csr_value,
    //                                     dA_tile_nnz_acc,
    //                                     dB_group_id,
    //                                     dB_bitmask,
    //                                     dB_group_ele_val,
    //                                     dB_spilled_row_hash_table_reverse_gmem,
    //                                     dB_tile_spilled_csrRowPtr,
    //                                     dB_tile_spilled_csrColInd,
    //                                     dB_tile_spilled_csrVal,
    //                                     dB_spilled_row_cnt_offset,
    //                                     dB_spilled_nnz_offset,
    //                                     dC_tile_spilled_csrColInd,
    //                                     dC_tile_spilled_csrVal
    //                                     );
    // }

    // printf("spgemm_compute_spilled success!\n");
    // spgemm_compute_1dthread_tcore<<<grid_2d, block_1d>>>(dB_bitmask, 
    //                                             dA_dense,
    //                                             dB_dense, 
    //                                             dB_group_id, 
    //                                             dB_spilled_row_hash_table_reverse_gmem,
    //                                             dB_group_ele_val,
    //                                             dB_spilled_row_cnt_offset,
    //                                             dB_spilled_nnz_offset,
    //                                             dB_tile_spilled_csrVal,                // output
    //                                             dB_tile_spilled_csrColInd,             // output
    //                                             dB_tile_spilled_csrRowPtr,             // output
    //                                             dA_tiled_csr_offset,
    //                                             dA_tiled_csr_column,
    //                                             dA_tiled_csr_value,
    //                                             dA_tile_nnz_acc,
    //                                             dA_tile_nnz,
    //                                             dC_output_group_idx,
    //                                             dC_final_result_gmem
    //                                             );

    // ValueType *d_probe;
    // CHECK_CUDA( hipMalloc((void**) &d_probe,     16 * 8 * 32 * sizeof(ValueType)) )

    spgemm_compute_1dthread_tcore_v2<<<grid_2d, block_1d>>>(
                                                dB_bitmask, 
                                                dB_group_id, 
                                                dB_spilled_row_hash_table_reverse_gmem,
                                                dB_group_ele_val,

                                                dB_spilled_row_cnt_offset,
                                                dB_spilled_nnz_offset,

                                                dB_tile_spilled_csrVal,                // output
                                                dB_tile_spilled_csrColInd,             // output
                                                dB_tile_spilled_csrRowPtr,             // output

                                                dA_tiled_csr_offset,
                                                dA_tiled_csr_column,
                                                dA_tiled_csr_value,
                                                dA_tile_nnz_acc,

                                                dC_output_group_idx,
                                                dC_group_value,
                                                d_multiplicand
                                                // d_probe
                                                );


    // ValueType *h_probe = (ValueType *)malloc(16 * 8 * 32 * sizeof(ValueType));
    // hipMemcpy(h_probe, d_probe, 16 * 8 * 32 * sizeof(ValueType), hipMemcpyDeviceToHost);
    // printf("group_indicator\n");
    // printMatrix(32, 32, h_probe, "group_indicator");

    // OutputType* hC_dense = (OutputType*)malloc(sizeof(OutputType)*m*n);
    // OutputType* dC_dense;
    // CHECK_CUDA( hipMalloc((void**) &dC_dense, m * n * sizeof(OutputType)) )
    // group2dense<<<grid_2d, block_1d>>>(dC_group_value, dC_dense, dC_output_group_idx, dC_bitmask);
    // hipMemcpy(hC_dense, dC_dense, SIZE_M * SIZE_N * sizeof(OutputType), hipMemcpyDeviceToHost);
    
    // printMatrixTile(32, 32, SIZE_K, hA_dense, "hA_dense");
    // printMatrixTile(256, 32, SIZE_K, hB_dense, "hB_dense");
    // printMatrixTile(32, 32, SIZE_N, hC_dense, "BitSparse result");

    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     printf("CUDA Error: %s\n", hipGetErrorString(err));
    // }

    float* hC_group_value = (float *)malloc(tileC_cnt * TILE_WIDTH * (OUTPUT_MAX_GROUP_NUM*4) * sizeof(float));
    hipMemcpy(hC_group_value, dC_group_value, tileC_cnt * TILE_WIDTH * (OUTPUT_MAX_GROUP_NUM*4) * sizeof(float), hipMemcpyDeviceToHost);
    printf("group_value\n");
    printMatrix(16, 32, hC_group_value, "hC_group_value", 6);

    hipFree(dC_group_value);
    // hipFree(d_multiplicand);
    // hipFree(dC_spilled_row_cnt);
    // hipFree(dC_spilled_nnz);
    // hipFree(dC_groupmask);

    float ms = 2;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);


    ValueType *hB_group_ele_val = (ValueType *)malloc(SIZE_K * SIZE_N / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType));
    hipMemcpy(hB_group_ele_val, dB_group_ele_val, SIZE_K * SIZE_N / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType), hipMemcpyDeviceToHost);
    printf("dB_group_value\n");
    printMatrix(32, 32, hB_group_ele_val, "group");

    int *hB_groupmask = (int*)malloc(tileB_cnt * MAX_GROUP_NUM * sizeof(int));
    hipMemcpy(hB_groupmask, dB_groupmask, tileB_cnt * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);
    printf("B_groupmask\n");
    printintMatrix_32(32, hB_groupmask, "B_groupmask");
    // int *hC_spilled_row_cnt = (int*)malloc(tileC_cnt * sizeof(int));
    // int *hC_spilled_nnz = (int*)malloc(tileC_cnt * sizeof(int));
    // hipMemcpy(hC_spilled_row_cnt, dC_spilled_row_cnt, tileC_cnt * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(hC_spilled_nnz, dC_spilled_nnz, tileC_cnt * sizeof(int), hipMemcpyDeviceToHost);

    return ms;

}

float timing_tSparse(int64_t &nnzA, int64_t &nnzB, float*& dA_dense, float *&dB_dense)
{
    // tSparse
    typedef typename cusp::array1d_view< thrust::device_ptr<float> > DeviceArray1dView;
    typedef cusp::array2d_view<DeviceArray1dView, cusp::row_major> DeviceArray2dView;

    // *NOTE* raw pointers must be wrapped with thrust::device_ptr!
    thrust::device_ptr<float> wrapped_device_A(dA_dense);
    // use array1d_view to represent the linear array data
    DeviceArray1dView linear_array_A(wrapped_device_A, wrapped_device_A + SIZE_M*SIZE_K);
    // use array2d_view to wrap the linear array
    DeviceArray2dView A_dense(SIZE_M, SIZE_K, SIZE_K, linear_array_A);

    thrust::device_ptr<float> wrapped_device_B(dB_dense);
    DeviceArray1dView linear_array_B(wrapped_device_B, wrapped_device_B + SIZE_K*SIZE_N);
    DeviceArray2dView B_dense(SIZE_K, SIZE_N, SIZE_N, linear_array_B);

    cusp::coo_matrix<int, float, cusp::host_memory> A_COO_h(SIZE_M, SIZE_K, nnzA);
    cusp::coo_matrix<int, float, cusp::host_memory> B_COO_h(SIZE_K, SIZE_N, nnzB);
    cusp::array2d<float, cusp::host_memory, cusp::row_major> A_dense_h(A_dense);
    cusp::array2d<float, cusp::host_memory, cusp::row_major> B_dense_h(B_dense);

    cusp::convert(A_dense_h, A_COO_h);
    cusp::convert(B_dense_h, B_COO_h);

    float tsparse_ms = time_spmmBMP_noTuple(A_COO_h, B_COO_h);

    return tsparse_ms;
}

float timing_tileSpgemm(int64_t &nnzA, int64_t &nnzB,
                        int *&dA_csr_offsets, 
                        int *&dA_csr_columns, 
                        float *&dA_csr_values,

                        int *&dB_csr_offsets, 
                        int *&dB_csr_columns, 
                        float *&dB_csr_values
                        )
{
    int *hA_csr_offsets = (int*)malloc(sizeof(int) * (SIZE_M + 1));
    int *hA_csr_columns = (int*)malloc(sizeof(int) * nnzA);
    float *hA_csr_values = (float*)malloc(sizeof(float) * nnzA);
    hipMemcpy(hA_csr_offsets, dA_csr_offsets, sizeof(int) * (SIZE_M + 1), hipMemcpyDeviceToHost);
    hipMemcpy(hA_csr_columns, dA_csr_columns, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
    hipMemcpy(hA_csr_values, dA_csr_values, sizeof(float) * nnzA, hipMemcpyDeviceToHost);

    int *hB_csr_offsets = (int*)malloc(sizeof(int) * (SIZE_K + 1));
    int *hB_csr_columns = (int*)malloc(sizeof(int) * nnzB);
    float *hB_csr_values = (float*)malloc(sizeof(float) * nnzB);
    hipMemcpy(hB_csr_offsets, dB_csr_offsets, sizeof(int) * (SIZE_K + 1), hipMemcpyDeviceToHost);
    hipMemcpy(hB_csr_columns, dB_csr_columns, sizeof(int) * nnzB, hipMemcpyDeviceToHost);
    hipMemcpy(hB_csr_values, dB_csr_values, sizeof(float) * nnzB, hipMemcpyDeviceToHost);

    // TileSpGEMM
	SMatrix *matrixA = (SMatrix *)malloc(sizeof(SMatrix));
	SMatrix *matrixB = (SMatrix *)malloc(sizeof(SMatrix));

    initialize_SMatrix(matrixA, SIZE_M, SIZE_K, nnzA, hA_csr_offsets, hA_csr_columns, hA_csr_values);
    initialize_SMatrix(matrixB, SIZE_K, SIZE_N, nnzB, hB_csr_offsets, hB_csr_columns, hB_csr_values);

    unsigned long long int nnzCub = 0;
    for (int i = 0; i < matrixA->nnz; i++)
    {
        int rowidx = matrixA->columnindex[i];
        nnzCub += matrixB->rowpointer[rowidx + 1] - matrixB->rowpointer[rowidx];
    }

    csr2tile_row_major(matrixA);
    csr2tile_col_major(matrixB);

    free(matrixA->rowpointer);
    free(matrixA->columnindex);
    free(matrixA->value);

    int blk_intersec_bitmask_len = ceil((double)matrixA->tilen / 32.0);
    double densityA = (double)matrixA->numtile / ((double)matrixA->tilem*(double)matrixA->tilen);
    double densityB = (double)matrixB->numtile / ((double)matrixB->tilem*(double)matrixB->tilen);

    long long int lengthA = (long long int) (matrixA->tilem) * (long long int)( blk_intersec_bitmask_len) ;
    unsigned int *blk_intersec_bitmask_A = (unsigned int *)malloc(lengthA* sizeof(unsigned int));
    memset(blk_intersec_bitmask_A, 0, lengthA * sizeof(unsigned int));
    for (int i = 0; i < matrixA->tilem; i++)
    {
        for (int j = matrixA->tile_ptr[i]; j < matrixA->tile_ptr[i + 1]; j++)
        {
            int idx = matrixA->tile_columnidx[j];
            unsigned int bitmask = 1;
            bitmask <<=  (31- (idx % 32));
            long long int pos = (long long int)i * (long long int)blk_intersec_bitmask_len + idx / 32;
            blk_intersec_bitmask_A[pos] |= bitmask;
        }
    }

    long long int lengthB = (long long int) (matrixB->tilen) * (long long int)(blk_intersec_bitmask_len) ;
    unsigned int *blk_intersec_bitmask_B = (unsigned int *)malloc(lengthB * sizeof(unsigned int));
    memset(blk_intersec_bitmask_B, 0, lengthB * sizeof(unsigned int));
    for (int i = 0; i < matrixB->tilen; i++)
    {
        for (int j = matrixB->csc_tile_ptr[i]; j < matrixB->csc_tile_ptr[i+1]; j++)
        {
            int idx = matrixB->csc_tile_rowidx[j];
            unsigned int bitmask = 0x1;
            bitmask <<= (31 - (idx % 32));
            long long int pos = (long long int)i * (long long int )blk_intersec_bitmask_len + idx / 32;
            blk_intersec_bitmask_B[pos] |= bitmask;
        }
    }

    // generate rowidx of blockA
    int *tile_rowidx_A = (int *)malloc (matrixA->numtile * sizeof(int ) );
    for (int i = 0; i < matrixA->tilem; i++)
    {
        for (int j = matrixA->tile_ptr[i]; j < matrixA->tile_ptr[i+1]; j++)
        {
            tile_rowidx_A[j] = i;
        }
    }

    SMatrix *matrixC = (SMatrix *)malloc(sizeof(SMatrix));
    
    struct timeval tv;
    unsigned long long int nnzC_computed;
    double compression_rate = 0;
    double time_tile = 0;
    double gflops_tile = 0;
    double time_step1 =0,time_step2 =0,time_step3 =0,time_malloc=0; 

    float tilespgemm_time = tilespgemm(matrixA,
               matrixB,
               matrixC,
               blk_intersec_bitmask_A,
               blk_intersec_bitmask_B,
               blk_intersec_bitmask_len,
               densityA,
               densityB,
               nnzCub,
               &nnzC_computed,
               &compression_rate,
               &time_tile,
               &gflops_tile,
               &time_step1,&time_step2,&time_step3,&time_malloc);

    return tilespgemm_time;
}

int cusparse_sparse2dense(int64_t &nnz, int *&d_csr_offsets, int *&d_csr_columns, float *&d_csr_values, float *&d_dense)
{
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, SIZE_M, SIZE_N, nnz,
                                      d_csr_offsets, d_csr_columns,
                                      d_csr_values, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, SIZE_M, SIZE_N, SIZE_N, d_dense,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSparseToDense_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseSparseToDense(handle, matA, matB,
                                          HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                          dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    return 0;
}


int main(int argc, char ** argv) 
{
    // using IndexType = int;
    // using ValueType = float;
    // using CSRHost = cusp::csr_matrix<IndexType,ValueType,cusp::host_memory>;
    // using CSRDev = cusp::csr_matrix<IndexType,ValueType,cusp::device_memory>;

    // hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);

    const int m = SIZE_M;
    const int k = SIZE_K;
	const int n = SIZE_N;

    dim3 grid2(SIZE_M/TILE_HEIGHT, SIZE_N/TILE_WIDTH, 1), block2(TILE_HEIGHT, 1, 1);

	InitValueType* hA_dense = (InitValueType*)malloc(sizeof(InitValueType)*m*k);
    InitValueType* hB_dense = (InitValueType*)malloc(sizeof(InitValueType)*k*n);
    float* hC_dense_float = (float*)malloc(sizeof(float)*m*n);
    fill_random(hA_dense, m, k, SPARSITY_A);
    fill_random(hB_dense, k, n, SPARSITY_B);
    // fill_random(hC_dense, m, n, SPARSITY);

    //--------------------------------------------------------------------------
    // basic ptrs
    InitValueType *dA_dense, *dA_csr_values;
    InitValueType *dB_dense, *dB_csr_values;
    int   *dA_csr_offsets, *dA_csr_columns;
    int   *dB_csr_offsets, *dB_csr_columns;
    int   *dC_csrOffsets, *dC_columns;
    OutputType *dC_values;

    //--------------------------------------------------------------------------
    // advanced ptrs
    // Matrix A
    int *dA_tiled_csr_offset, *dA_tiled_csr_column;
    int *dA_tile_nnz_acc, *dA_tile_nnz, *dA_tile_row_nnz;
    ValueType *dA_tiled_csr_value;

    // Matrix B
    ValueType *dB_group_ele_val;
    int *dB_group_id, *dB_spilled_row_cnt, *dB_spilled_nnz;
    int *dB_spilled_row_hash_table_gmem, *dB_spilled_row_hash_table_reverse_gmem;
    int *dB_group_ele_ind;
    BitMaskType *dB_bitmask, *dB_groupmask;
    ValueType *dB_tile_spilled_csrVal;
    int *dB_tile_spilled_csrColInd, *dB_tile_spilled_csrRowPtr;
    int *dB_spilled_nnz_offset, *dB_spilled_row_cnt_offset;

    // Matrix C
    float *dC_group_value;
    int *dC_bitmask;
    int *dC_output_group_idx;

    //--------------------------------------------------------------------------
    // Constants
    int tileA_cnt = (SIZE_M/TILE_HEIGHT)*(SIZE_K/SPLIT_K);
    int tileB_cnt = SIZE_K * SIZE_N / SPLIT_K / TILE_WIDTH;
    int tileB_x_cnt = SIZE_N / TILE_WIDTH;
    int tileB_y_cnt = SIZE_K / SPLIT_K;
    int tileC_cnt = SIZE_M * SIZE_N / TILE_HEIGHT / TILE_WIDTH;

    //--------------------------------------------------------------------------
    // basic allocation
    CHECK_CUDA( hipMalloc((void**) &dA_dense,          m * k * sizeof(InitValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dA_csr_offsets,   (m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_dense,          k * n * sizeof(InitValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_csr_offsets,   (k + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,    (m + 1) * sizeof(int)) )

    //--------------------------------------------------------------------------
    // advanced allocation
    CHECK_CUDA( hipMalloc((void**) &dB_bitmask,        k * n / TILE_WIDTH * sizeof(BitMaskType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_groupmask,      tileB_cnt * MAX_GROUP_NUM * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_ele_ind,  k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_ele_val,  k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_group_id,       k * n / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_cnt,tileB_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_nnz,    tileB_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_hash_table_gmem, tileB_cnt * SPLIT_K * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_hash_table_reverse_gmem, tileB_cnt * SPLIT_K * sizeof(int)) )

    
    CHECK_CUDA( hipMemcpy(dA_dense, hA_dense, m * k * sizeof(InitValueType), hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_dense, hB_dense, k * n * sizeof(InitValueType), hipMemcpyHostToDevice) )

    //--------------------------------------------------------------------------
    // Format conversion
    dim3 grid_for_convert_A(SIZE_M/32, SIZE_K/32, 1), grid_for_convert_B(SIZE_K/32, SIZE_N/32, 1);
    dim3 block_for_convert(32, 32, 1);
    half *dA_dense_half, *dB_dense_half;
    CHECK_CUDA( hipMalloc((void**) &dA_dense_half,  SIZE_M * SIZE_K * sizeof(half)) )
    CHECK_CUDA( hipMalloc((void**) &dB_dense_half,  SIZE_K * SIZE_N * sizeof(half)) )
    
    format_convert<<<grid_for_convert_A, block_for_convert>>>(dA_dense, dA_dense_half);
    format_convert<<<grid_for_convert_B, block_for_convert>>>(dB_dense, dB_dense_half);

    ValueType *dA_dense_int8, *dB_dense_int8;
    CHECK_CUDA( hipMalloc((void**) &dA_dense_int8,  SIZE_M * SIZE_K * sizeof(ValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_dense_int8,  SIZE_K * SIZE_N * sizeof(ValueType)) )
    
    format_convert<<<grid_for_convert_A, block_for_convert>>>(dA_dense, dA_dense_int8);
    format_convert<<<grid_for_convert_B, block_for_convert>>>(dB_dense, dB_dense_int8);

    //--------------------------------------------------------------------------
    // Transform dense to CSR
    int64_t nnzA, nnzB, nnzC;
    dense2CSR(m, k, dA_dense, dA_csr_values, dA_csr_offsets, dA_csr_columns, nnzA);
    dense2CSR(k, n, dB_dense, dB_csr_values, dB_csr_offsets, dB_csr_columns, nnzB);
    std::cout << "nnzA: " << nnzA << ",  nnzB: " << nnzB << std::endl;

    //--------------------------------------------------------------------------
    // Timing

    // bitSpgemm
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    // bitspgemm_prepare(nnzA, nnzB, nnzC, dA_csr_values, dA_csr_offsets, dA_csr_columns, dA_tiled_csr_offset,
    //                     dA_tiled_csr_column, dA_tiled_csr_value, dA_tile_nnz_acc, dA_tile_nnz, dA_tile_row_nnz, 
    //                     dB_dense, dB_bitmask, dB_groupmask, dB_group_ele_val, dB_group_id, 
    //                     dB_spilled_row_cnt, dB_spilled_nnz, dB_spilled_row_hash_table_gmem, dB_spilled_row_hash_table_reverse_gmem,
    //                     dB_tile_spilled_csrVal, dB_tile_spilled_csrColInd, dB_tile_spilled_csrRowPtr, dB_spilled_nnz_offset, dB_spilled_row_cnt_offset);

    // float ms = timing_bitspgemm(nnzA, nnzB, nnzC, dA_tiled_csr_offset, dA_tiled_csr_column, dA_tiled_csr_value, dA_tile_nnz_acc, dA_tile_nnz, dA_tile_row_nnz,
    //                     dB_dense, dB_bitmask, dB_groupmask, dB_group_ele_val, dB_group_id, 
    //                     dB_spilled_row_cnt, dB_spilled_nnz, dB_spilled_row_hash_table_gmem, dB_spilled_row_hash_table_reverse_gmem,
    //                     dB_tile_spilled_csrVal, dB_tile_spilled_csrColInd, dB_tile_spilled_csrRowPtr, dB_spilled_nnz_offset, dB_spilled_row_cnt_offset,
    //                     dC_output_group_idx, dC_bitmask, dC_group_value);

    printf("Transform CSR to tiled CSR\n");
    CHECK_CUDA( hipMalloc((void**) &dA_tile_nnz,         sizeof(int) * tileA_cnt) )
    CHECK_CUDA( hipMalloc((void**) &dA_tile_nnz_acc,     sizeof(int) * (tileA_cnt+1)) )
    CHECK_CUDA( hipMalloc((void**) &dA_tile_row_nnz,     sizeof(int) * SIZE_M * SIZE_K / SPLIT_K) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_offset, sizeof(int) * tileA_cnt * (TILE_HEIGHT+1)) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_column, sizeof(int) * nnzA) )
    CHECK_CUDA( hipMalloc((void**) &dA_tiled_csr_value,  sizeof(ValueType) * nnzA) )

    csr2tiledcsr<<<1, 1>>>(tileA_cnt, 
                            nnzA, 
                            dA_csr_offsets, 
                            dA_csr_columns, 
                            dA_csr_values,
                            dA_tiled_csr_offset,
                            dA_tiled_csr_column,
                            dA_tiled_csr_value,
                            dA_tile_nnz_acc,
                            dA_tile_nnz,
                            dA_tile_row_nnz
                            );

    // int *hA_tiled_csr_offset = (int*)malloc(sizeof(int) * tileA_cnt * (TILE_HEIGHT+1));
    // int *hA_tiled_csr_column = (int*)malloc(sizeof(int) * nnzA);
    // ValueType *hA_tiled_csr_value = (ValueType*)malloc(sizeof(ValueType) * nnzA);
    // hipMemcpy(hA_tiled_csr_offset, dA_tiled_csr_offset, sizeof(int) * tileA_cnt * (TILE_HEIGHT+1), hipMemcpyDeviceToHost);
    // hipMemcpy(hA_tiled_csr_column, dA_tiled_csr_column, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
    // hipMemcpy(hA_tiled_csr_value, dA_tiled_csr_value, sizeof(ValueType) * nnzA, hipMemcpyDeviceToHost);

    dim3 grid1(tileB_x_cnt, tileB_y_cnt, 1), block1(SPLIT_K, 1, 1);
    printf("Matrix B dense2bitmask...\n");
    dense2bitmask<<<grid1, block1>>>(dB_dense, dB_bitmask);

    int *dB_nnz;
    CHECK_CUDA( hipMalloc((void**) &dB_nnz, sizeof(int) * 1) )
    int *hB_nnz = (int*)malloc(sizeof(int));

    printf("\nMatrix B generate groups...\n");
    generate_groups<<<grid1, block1>>>(dB_bitmask,                            // input
                                     dB_groupmask,                          // output, for visualization
                                    //  dB_group_ele_ind,                      // output, not necessary
                                     dB_group_ele_val,                      // output
                                     dB_dense,                              // input
                                     dB_group_id,                           // output
                                     dB_spilled_row_cnt,                    // output
                                     dB_spilled_nnz,
                                    //  dB_tile_spilled_csrVal,                // output
                                    //  dB_tile_spilled_csrColInd,             // output
                                    //  dB_tile_spilled_csrRowPtr,             // output
                                     dB_spilled_row_hash_table_gmem,
                                     dB_spilled_row_hash_table_reverse_gmem,   // output
                                     dB_nnz
                                     );

    hipMemcpy(hB_nnz, dB_nnz, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "nnz of B in groups: " << *hB_nnz << std::endl; 

    ValueType *hB_group_ele_val = (ValueType *)malloc(k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType));
    hipMemcpy(hB_group_ele_val, dB_group_ele_val, k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType), hipMemcpyDeviceToHost);
    printf("dB_group_value\n");
    printMatrix(32, 32, hB_group_ele_val, "group");

    int *hB_groupmask = (int*)malloc(tileB_cnt * MAX_GROUP_NUM * sizeof(int));
    hipMemcpy(hB_groupmask, dB_groupmask, tileB_cnt * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);
    printf("B_groupmask\n");
    printintMatrix_32(32, hB_groupmask, "B_groupmask");
    

    int *hB_spilled_nnz = (int*)malloc(tileB_cnt * sizeof(int));
    int *hB_spilled_row_cnt = (int*)malloc(tileB_cnt * sizeof(int));
    hipMemcpy(hB_spilled_nnz, dB_spilled_nnz, tileB_cnt * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hB_spilled_row_cnt, dB_spilled_row_cnt, tileB_cnt * sizeof(int), hipMemcpyDeviceToHost);

    int nnz_cnt = 0;
    int row_cnt = 0;
    int *hB_spilled_nnz_offset = (int*)malloc(tileB_cnt * sizeof(int));
    int *hB_spilled_row_cnt_offset = (int*)malloc(tileB_cnt * sizeof(int));
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_nnz_offset,     tileB_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_spilled_row_cnt_offset,  tileB_cnt * sizeof(int)) )
    for (int i = 0; i < tileB_cnt; i++)
    {
        hB_spilled_nnz_offset[i] = nnz_cnt;
        hB_spilled_row_cnt_offset[i] = row_cnt;
        nnz_cnt += hB_spilled_nnz[i];
        row_cnt += hB_spilled_row_cnt[i];
    }
    hipMemcpy(dB_spilled_nnz_offset, hB_spilled_nnz_offset, tileB_cnt * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB_spilled_row_cnt_offset, hB_spilled_row_cnt_offset, tileB_cnt * sizeof(int), hipMemcpyHostToDevice);

    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrVal,     nnz_cnt * sizeof(ValueType)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrColInd,  nnz_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_tile_spilled_csrRowPtr,  row_cnt * sizeof(int)) )

    generate_spilled_csr<<<grid1, block1>>>(dB_bitmask,                            // input
                                     dB_dense,                              // input
                                     dB_group_id,                           // output
                                     dB_spilled_row_cnt,                    // output
                                     dB_spilled_nnz,
                                     dB_spilled_row_cnt_offset,
                                     dB_spilled_nnz_offset,
                                     dB_tile_spilled_csrVal,                // output
                                     dB_tile_spilled_csrColInd,             // output
                                     dB_tile_spilled_csrRowPtr,             // output
                                     dB_spilled_row_hash_table_gmem,
                                     dB_spilled_row_hash_table_reverse_gmem // output
                                     );

    std::cout << "Total nnz: " << nnz_cnt << "  Total row_cnt: " << row_cnt << std::endl;
    CHECK_CUDA(hipDeviceSynchronize())

    half *d_multiplicand;
    CHECK_CUDA( hipMalloc((void**) &d_multiplicand,  8 * 16 * sizeof(half)) )
    half *h_multiplicand = (half*)malloc(8 * 16 * sizeof(half));
    initialize_multiplicand(h_multiplicand);
    hipMemcpy(d_multiplicand, h_multiplicand, 8 * 16 * sizeof(half), hipMemcpyHostToDevice);


    BitMaskType *dC_groupmask;
    int *dC_spilled_row_cnt, *dC_spilled_nnz;
    int *dC_spilled_row_row_idx, *dC_spilled_row_tile_idx;
    CHECK_CUDA( hipMalloc((void**) &dC_group_value,  tileC_cnt * (OUTPUT_MAX_GROUP_NUM*4) * TILE_WIDTH * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_bitmask,  SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_groupmask,  tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_cnt,  tileC_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz,  tileC_cnt * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_output_group_idx,  SIZE_M * SIZE_N / TILE_WIDTH * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_row_idx,  MAX_SPILLED_ROW_CNT_C * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_tile_idx,  MAX_SPILLED_ROW_CNT_C * sizeof(int)) )

    int *dC_spilled_row_buffersize, *dC_spilled_nnz_buffersize;
    int *dC_spilled_nnz_offset, *dC_spilled_row_cnt_offset;
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz_offset,     (tileC_cnt + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_cnt_offset,  (tileC_cnt + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_row_buffersize,  sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_spilled_nnz_buffersize,  sizeof(int)) )

    int *dC_nnz;
    CHECK_CUDA( hipMalloc((void**) &dC_nnz, sizeof(int) * 1) )
    int *hC_nnz = (int*)malloc(sizeof(int));
    
    dim3 grid_2d(SIZE_N/TILE_WIDTH, SIZE_M/TILE_HEIGHT, 1), block_1d(TILE_HEIGHT, 1, 1);
    hipEventRecord(start);
    pre_spgemm<<<grid_2d, block_1d>>>(dB_bitmask, 
                                      dC_spilled_row_cnt, 
                                      dC_spilled_nnz, 
                                      dA_tiled_csr_offset,
                                      dA_tiled_csr_column,  
                                      dA_tile_nnz_acc, 
                                      dC_output_group_idx,
                                      dC_bitmask,
                                      dC_groupmask,
                                      dC_spilled_row_row_idx,
                                      dC_spilled_row_tile_idx,
                                      dC_spilled_row_cnt_offset,
                                      dC_spilled_nnz_offset,
                                      dC_spilled_row_buffersize,
                                      dC_spilled_nnz_buffersize,
                                      dC_nnz
                                      );


    // BitMaskType* hC_groupmask = (BitMaskType*)malloc(tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType));
    // hipMemcpy(hC_groupmask, dC_groupmask, tileC_cnt * OUTPUT_MAX_GROUP_NUM * sizeof(BitMaskType), hipMemcpyDeviceToHost);
    // printf("\n hC_groupmask: %d\n", hC_groupmask[0]);
    // printintMatrix_32(16, hC_groupmask, "hC_groupmask");

    spgemm_compute_1dthread_tcore_v2<<<grid_2d, block_1d>>>(
                                                dB_bitmask, 
                                                dB_group_id, 
                                                dB_spilled_row_hash_table_reverse_gmem,
                                                dB_group_ele_val,

                                                dB_spilled_row_cnt_offset,
                                                dB_spilled_nnz_offset,

                                                dB_tile_spilled_csrVal,                // output
                                                dB_tile_spilled_csrColInd,             // output
                                                dB_tile_spilled_csrRowPtr,             // output

                                                dA_tiled_csr_offset,
                                                dA_tiled_csr_column,
                                                dA_tiled_csr_value,
                                                dA_tile_nnz_acc,

                                                dC_output_group_idx,
                                                dC_group_value,
                                                d_multiplicand
                                                // d_probe
                                                );
    hipEventRecord(end);
    hipEventSynchronize(end);

    float* hC_group_value = (float *)malloc(tileC_cnt * TILE_WIDTH * (OUTPUT_MAX_GROUP_NUM*4) * sizeof(float));
    hipMemcpy(hC_group_value, dC_group_value, tileC_cnt * TILE_WIDTH * (OUTPUT_MAX_GROUP_NUM*4) * sizeof(float), hipMemcpyDeviceToHost);
    printf("group_value\n");
    printMatrix(16, 32, hC_group_value, "hC_group_value", 6);

    float ms = 0;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    // cusparse: spgemm
    float cusparse_ms = timing_cusparse_spgemm(nnzA, nnzB, nnzC, dA_csr_offsets, dA_csr_columns, dA_csr_values, 
                                        dB_csr_offsets, dB_csr_columns, dB_csr_values,
                                        dC_csrOffsets, dC_columns, dC_values);
    // cusparse: spmm
    float cusparse_spmm_ms = timing_cusparse_spmm_csr(nnzA, dA_csr_offsets, dA_csr_columns, dA_csr_values, dB_dense);
    // cusparseLt
    float cusparseLt_ms = timing_cusparseLt(dA_dense_int8, dB_dense_int8);
    // tSparse
    float tsparse_ms = timing_tSparse(nnzA, nnzB, dA_dense, dB_dense);
    // TileSpGEMM
    float tilespgemm_time = timing_tileSpgemm(nnzA, nnzB, dA_csr_offsets, dA_csr_columns, dA_csr_values, dB_csr_offsets, dB_csr_columns, dB_csr_values);


    // ValueType *hB_group_ele_val = (ValueType *)malloc(SIZE_K * SIZE_N / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType));
    // hipMemcpy(hB_group_ele_val, dB_group_ele_val, SIZE_K * SIZE_N / SPLIT_K * MAX_GROUP_NUM * sizeof(ValueType), hipMemcpyDeviceToHost);
    // printf("dB_group_value\n");
    // printMatrix(32, 32, hB_group_ele_val, "group");

    // int *hB_groupmask = (int*)malloc(tileB_cnt * MAX_GROUP_NUM * sizeof(int));
    // hipMemcpy(hB_groupmask, dB_groupmask, tileB_cnt * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);
    // printf("B_groupmask\n");
    // printintMatrix_32(32, hB_groupmask, "B_groupmask");

    // float *dC_dense_float;
    // CHECK_CUDA( hipMalloc((void**) &dC_dense_float, SIZE_M * SIZE_N * sizeof(float)))
    // cusparse_sparse2dense(nnzC, dC_csrOffsets, dC_columns, dC_values, dC_dense_float);
    // CHECK_CUDA( hipMemcpy(hC_dense_float, dC_dense_float, SIZE_M * SIZE_N * sizeof(float), hipMemcpyDeviceToHost) )
    // // printMatrixTile(16, 32, SIZE_N, hC_dense_float, "Mat C ground truth (tile)");

    // float *dC_group_float;
    // CHECK_CUDA( hipMalloc((void**) &dC_group_float,  tileC_cnt * OUTPUT_MAX_GROUP_NUM * TILE_WIDTH * sizeof(float)) )
    // dim3 grid_2d(SIZE_N/TILE_WIDTH, SIZE_M/TILE_HEIGHT, 1), block_1d(TILE_HEIGHT, 1, 1);
    // dense2group_from_idx<<<grid_2d, block_1d>>>(dC_dense_float, dC_group_float, dC_output_group_idx, dC_bitmask);
    // float *hC_group_float = (float *)malloc(tileC_cnt * OUTPUT_MAX_GROUP_NUM * TILE_WIDTH * sizeof(float));
    // CHECK_CUDA( hipMemcpy(hC_group_float, dC_group_float, tileC_cnt * OUTPUT_MAX_GROUP_NUM * TILE_WIDTH * sizeof(float), hipMemcpyDeviceToHost) )
    // printf("\n\nMat C group rebuild from ground truth\n");
    // printMatrix(16, 32, hC_group_float, "Mat C group rebuild from ground truth", 6);

    // printf("matrixA-nnz: %d\n", matrixA->nnz);

    printf("bitSparse elapsed time:          %fms\n", ms);
    printf("cusparse-SpGEMM elapsed time:    %fms\n", cusparse_ms);
    printf("cusparse-SpMM elapsed time:      %fms\n", cusparse_spmm_ms);
    printf("cusparseLt elapsed time:         %fms\n", cusparseLt_ms);
    printf("tSparse elpased time:            %fms\n", tsparse_ms);
    printf("TileSpGEMM elpased time:         %fms\n", tilespgemm_time);


    // printf("\nC_sparsity: %f, nnz_C: %d\n", 1.0 - float(C_nnz1)/SIZE_M/SIZE_N, C_nnz1);
    
    // print MatA's information
    if (PRINT_MAT_A_INFO)
    {

        int *hA_tiled_csr_offset = (int*)malloc(sizeof(int) * tileA_cnt * (TILE_HEIGHT+1));
        int *hA_tiled_csr_column = (int*)malloc(sizeof(int) * nnzA);
        float *hA_tiled_csr_value = (float*)malloc(sizeof(float) * nnzA);
        int *hA_tile_nnz = (int*)malloc(sizeof(int) * tileA_cnt);
        int *hA_tile_nnz_acc = (int*)malloc(sizeof(int) * (tileA_cnt+1));

        hipMemcpy(hA_tiled_csr_value, dA_tiled_csr_value, sizeof(float) * nnzA, hipMemcpyDeviceToHost);
        hipMemcpy(hA_tiled_csr_column, dA_tiled_csr_column, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
        hipMemcpy(hA_tiled_csr_offset, dA_tiled_csr_offset, sizeof(int) * tileA_cnt * (TILE_HEIGHT+1), hipMemcpyDeviceToHost);
        hipMemcpy(hA_tile_nnz, dA_tile_nnz, sizeof(int) * tileA_cnt, hipMemcpyDeviceToHost);
        hipMemcpy(hA_tile_nnz_acc, dA_tile_nnz_acc, sizeof(int) * (tileA_cnt + 1), hipMemcpyDeviceToHost);

        printf("nnzA: %ld\n", nnzA);
        for (int i = 0; i < tileA_cnt+1; i++)
        {
            printf("hA_tile_nnz_acc: %d\n", hA_tile_nnz_acc[i]);
        }

        for (int i = 0; i < nnzA; i++)
        {
            printf("hA_tiled_csr_value: %f\n", hA_tiled_csr_value[i]);
        }

        for (int i = 0; i < nnzA; i++)
        {
            printf("hA_tiled_csr_column: %d\n", hA_tiled_csr_column[i]);
        }

        for (int i = 0; i < tileA_cnt * (TILE_HEIGHT+1); i++)
        {
            printf("hA_tiled_csr_offset: %d\n", hA_tiled_csr_offset[i]);
        }
    }

    if (PRINT_MAT_B_INFO)
    {
        int *hB_group_id = (int*)malloc(sizeof(int) * k * n / TILE_WIDTH);
        hipMemcpy(hB_group_id, dB_group_id, sizeof(int) * k * n / TILE_WIDTH, hipMemcpyDeviceToHost);

        for (int i = 0; i < SPLIT_K; i++)
        {
            std::cout << "hB_group_id: " << hB_group_id[i] << std::endl;
        }

        // int *hB_tile_spilled_csrRowPtr = (int*)

        int *hB_spilled_row_cnt = (int*)malloc(k * n / SPLIT_K / TILE_WIDTH * sizeof(int));
        hipMemcpy(hB_spilled_row_cnt, dB_spilled_row_cnt, 
                k * n / SPLIT_K / TILE_WIDTH * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < k * n / SPLIT_K / TILE_WIDTH; i++)
        {
            std::cout << "hB_spilled_row_cnt: " << hB_spilled_row_cnt[i] << std::endl;
        }

        int *hB_spilled_row_hash_table_reverse_gmem = (int*)malloc(k * n / SPLIT_K / TILE_WIDTH * SPLIT_K * sizeof(int));
        hipMemcpy(hB_spilled_row_hash_table_reverse_gmem, dB_spilled_row_hash_table_reverse_gmem, 
                k * n / SPLIT_K / TILE_WIDTH * SPLIT_K * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < k * n / SPLIT_K / TILE_WIDTH * SPLIT_K; i++)
        {
            std::cout << "hB_spilled_row_hash_table_reverse_gmem-- " << i%SPLIT_K << ": " << hB_spilled_row_hash_table_reverse_gmem[i] << std::endl;
        }

        if (TILE_WIDTH == 64)
        {
            unsigned long long int *hB_bitmask = (unsigned long long int*)malloc(sizeof(unsigned long long int)*k*n/64);
            hipMemcpy(hB_bitmask, dB_bitmask, k * n / 64 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
            printlongintMatrix(k, hB_bitmask, "B_bitmask");
        }
        else if (TILE_WIDTH == 32)
        {
            int *hB_bitmask = (int*)malloc(sizeof(int) * k * n / 32);
            hipMemcpy(hB_bitmask, dB_bitmask, k * n / 32 * sizeof(int), hipMemcpyDeviceToHost);
            printintMatrix_32(k, hB_bitmask, "B_bitmask");
        }

        // int *hB_spilled_nnz = (int*)malloc(2 * sizeof(int));
        // hipMemcpy(hB_spilled_nnz, dB_spilled_nnz, 2 * sizeof(int), hipMemcpyDeviceToHost);
        // int *hB_tile_spilled_csrColInd = (int*)malloc(hB_spilled_nnz[0] * sizeof(int));
        // hipMemcpyFromSymbol(hB_tile_spilled_csrColInd, HIP_SYMBOL(dB_tile_spilled_csrColInd[0]), hB_spilled_nnz[0] * sizeof(int), 0, hipMemcpyDeviceToHost);

        // for (int i = 0; i < hB_spilled_nnz[0]; i++)
        // {
        //     std::cout << "hB_tile_spilled_csrColInd: " << hB_tile_spilled_csrColInd[i] << std::endl;
        // }

    }


    // if (TILE_WIDTH == 64)
    // {
    //     unsigned long long int *hB_groupmask = 
    //     (unsigned long long int*)malloc(k * n / SPLIT_K / TILE_WIDTH * MAX_GROUP_NUM * sizeof(unsigned long long int));
    //     hipMemcpy(hB_groupmask, dB_groupmask, k * n / SPLIT_K / TILE_WIDTH * MAX_GROUP_NUM * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    // }
    // else if (TILE_WIDTH == 32)
    // {
    //     int *hB_groupmask = (int*)malloc(k * n / SPLIT_K / TILE_WIDTH * MAX_GROUP_NUM * sizeof(int));
    //     hipMemcpy(hB_groupmask, dB_groupmask, k * n / SPLIT_K / TILE_WIDTH * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);
    //     printintMatrix_32(16, hB_groupmask, "B_groupmask");

    //     std::cout << "A random number: " << rand() % 100 << std::endl;
    //     int *hB_group_ele_ind = (int*)malloc(k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(int));
    //     hipMemcpy(hB_group_ele_ind, dB_group_ele_ind, k * n / SPLIT_K * MAX_GROUP_NUM * sizeof(int), hipMemcpyDeviceToHost);

    // }
    
    // size_t *size;
    // hipDeviceGetLimit(size, hipLimitMallocHeapSize);
    // std::cout << "HeapSize: " << *size << std::endl;

    // free(dB)

    // std::cout << "Input matrix A has shape (" << A.num_rows << "," << A.num_cols << ") and " << A.num_entries << " entries" << "\n";
    // std::cout << "             B has shape (" << B.num_rows << "," << B.num_cols << ") and " << B.num_entries << " entries" << "\n\n";

}